#include "hip/hip_runtime.h"
#include "../include/simulation_time.cuh"
#include "../include/cuda_utils/cuda_launcher.cuh"


CUDA_CALLABLE_MEMBER SimulationTime::SimulationTime() {

}
CUDA_CALLABLE_MEMBER SimulationTime::SimulationTime(real *startTime, real *endTime, real *dt) :
                                startTime(startTime), endTime(endTime), dt(dt) {

}

CUDA_CALLABLE_MEMBER SimulationTime::~SimulationTime() {

}

CUDA_CALLABLE_MEMBER void SimulationTime::set(real *dt, real *startTime, real *endTime, real *currentTime,
                                              real *dt_max) {
    this->dt = dt;
    this->startTime = startTime;
    this->endTime = endTime;
    this->currentTime = currentTime;
    this->dt_max = dt_max;
}

namespace SimulationTimeNS {
    namespace Kernel {
        __global__ void set(SimulationTime *simulationTime, real *dt, real *startTime, real *endTime,
                            real *currentTime, real *dt_max) {
            simulationTime->set(dt, startTime, endTime, currentTime, dt_max);
        }

        namespace Launch {
            void set(SimulationTime *simulationTime, real *dt, real *startTime, real *endTime,
                     real *currentTime, real *dt_max) {
                ExecutionPolicy executionPolicy(1, 1);
                cuda::launch(false, executionPolicy, ::SimulationTimeNS::Kernel::set, simulationTime, dt, startTime,
                             endTime, currentTime, dt_max);
            }
        }
    }
}