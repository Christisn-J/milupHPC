#include "hip/hip_runtime.h"
#include "../../include/mfv/riemann_fluxes.cuh"
#include "../include/cuda_utils/cuda_launcher.cuh"

namespace MFV {

    namespace Compute {
        __device__ void effectiveFace(real Aij[DIM], int i, int ip, int *interactions, Particles *particles){
            // search neighbor i in interactions[] of ip
            int d, ij;
            for(ij=0; ij<particles->noi[ip]; ij++){
                if (interactions[ij+ip*MAX_NUM_INTERACTIONS] == i) break;
            }

            Aij[0] = 1./particles->omega[i]*particles->psix[ip]
                    - 1./particles->omega[ip]*particles->psix[ij+ip*MAX_NUM_INTERACTIONS];
#if DIM > 1
            Aij[1] = 1./particles->omega[i]*particles->psiy[ip]
                    - 1./particles->omega[ip]*particles->psiy[ij+ip*MAX_NUM_INTERACTIONS];
#if DIM ==3
            Aij[0] = 1./particles->omega[i]*particles->psiz[ip]
                    - 1./particles->omega[ip]*particles->psiz[ij+ip*MAX_NUM_INTERACTIONS];
#endif
#endif
        }

        __device__ void gradient(real grad[DIM], real *f, int i, int *interactions, int noi, Particles *particles){
            int d, j, ip;
#pragma unroll
            for(d=0; d<DIM; d++){
                grad[d] = 0.;
            }

            for (j = 0; j < noi; j++) {
                ip = interactions[i * MAX_NUM_INTERACTIONS + j];

                grad[0] += (f[ip] - f[i]) * particles->psix[ip];
#if DIM > 1
                grad[1] += (f[ip] - f[i]) * particles->psiy[ip];
#if DIM == 3
                grad[2] += (f[ip] - f[i]) * particles->psiz[ip];
#endif
#endif
            }
        }

    }

    namespace Kernel {

        __global__ void riemannFluxes(Particles *particles, RiemannSolver riemannSolver, int *interactions, int numParticles){
            int i, j, inc, ip, noi;
            real x, rho, vx, P, vFrame[DIM], rhoGrad[DIM], vxGrad[DIM], pGrad[DIM]; // containers for particle i quantities
            real Aij[DIM]; // effective face of the interface i -> j
#if DIM > 1
            real y, vyGrad[DIM];
#if DIM == 3
            real z, vzGrad[DIM];
#endif
#endif
            /// main loop over particles
            inc = blockDim.x * gridDim.x;
            for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
                x = particles->x[i];
#if DIM > 1
                y = particles->y[i];
#if DIM == 3
                z = particles->z[i];
#endif
#endif
                noi = particles->noi[i];

                /// estimate gradients of particle i
                ::MFV::Compute::gradient(rhoGrad, particles->rho, i, interactions, noi, particles);
                ::MFV::Compute::gradient(vxGrad, particles->vx, i, interactions, noi, particles);
#if DIM > 1
                ::MFV::Compute::gradient(vyGrad, particles->vy, i, interactions, noi, particles);
#if DIM == 3
                ::MFV::Compute::gradient(vzGrad, particles->vz, i, interactions, noi, particles);
#endif
#endif
                ::MFV::Compute::gradient(pGrad, particles->p, i, interactions, noi, particles);

                /// loop over nearest neighbors
                for (j = 0; j < noi; j++) {
                    ip = interactions[i * MAX_NUM_INTERACTIONS + j];

                    ::MFV::Compute::effectiveFace(Aij, i, ip, interactions, particles);
                }


            }
        }

        namespace Launch {
            real riemannFluxes(Particles *particles, RiemannSolver riemannSolver, int *interactions, int numParticles) {
                ExecutionPolicy executionPolicy;
                return cuda::launch(true, executionPolicy, ::MFV::Kernel::riemannFluxes, particles, riemannSolver, interactions, numParticles);
            }
        }
    }
}