#include "hip/hip_runtime.h"
#include "../../include/subdomain_key_tree/tree.cuh"
#include "../../include/cuda_utils/cuda_launcher.cuh"

/*void launchBuildTreeKernel(Foo *foo) {
    ExecutionPolicy executionPolicy(1, 1);
    hipLaunchByPtr(false, executionPolicy, testKernel, foo);
    //testKernel<<<1, 1>>>(foo);
}*/

CUDA_CALLABLE_MEMBER Tree::Tree() {

}

CUDA_CALLABLE_MEMBER Tree::Tree(integer *count, integer *start, integer *child, integer *sorted, integer *index,
                                real *minX, real *maxX) : count(count), start(start), child(child), sorted(sorted),
                                index(index), minX(minX), maxX(maxX) {

}
CUDA_CALLABLE_MEMBER void Tree::set(integer *count, integer *start, integer *child, integer *sorted,
                                        integer *index, real *minX, real *maxX) {
    this->count = count;
    this->start = start;
    this->child = child;
    this->sorted = sorted;
    this->index = index;
    this->minX = minX;
    this->maxX = maxX;
}

#if DIM > 1
CUDA_CALLABLE_MEMBER Tree::Tree(integer *count, integer *start, integer *child, integer *sorted, integer *index,
                                real *minX, real *maxX, real *minY, real *maxY) : count(count), start(start),
                                child(child), sorted(sorted), index(index), minX(minX), maxX(maxX), minY(minY),
                                maxY(maxY) {

}
CUDA_CALLABLE_MEMBER void Tree::set(integer *count, integer *start, integer *child, integer *sorted,
                                        integer *index, real *minX, real *maxX, real *minY, real *maxY) {
    this->count = count;
    this->start = start;
    this->child = child;
    this->sorted = sorted;
    this->index = index;
    this->minX = minX;
    this->maxX = maxX;
    this->minY = minY;
    this->maxY = maxY;
}

#if DIM == 3
CUDA_CALLABLE_MEMBER Tree::Tree(integer *count, integer *start, integer *child, integer *sorted, integer *index,
                                real *minX, real *maxX, real *minY, real *maxY, real *minZ, real *maxZ) : count(count),
                                start(start), child(child), sorted(sorted), index(index), minX(minX), maxX(maxX),
                                minY(minY), maxY(maxY), minZ(minZ), maxZ(maxZ) {

}
CUDA_CALLABLE_MEMBER void Tree::set(integer *count, integer *start, integer *child, integer *sorted,
                                        integer *index, real *minX, real *maxX, real *minY, real *maxY,
                                        real *minZ, real *maxZ) {
    this->count = count;
    this->start = start;
    this->child = child;
    this->sorted = sorted;
    this->index = index;
    this->minX = minX;
    this->maxX = maxX;
    this->minY = minY;
    this->maxY = maxY;
    this->minZ = minZ;
    this->maxZ = maxZ;
}
#endif
#endif

CUDA_CALLABLE_MEMBER void Tree::reset(integer index, integer n) {
    #pragma unroll 8
    for (integer i=0; i<POW_DIM; i++) {
        child[index * POW_DIM + i] = -1;
    }

    if (index < n) {
        count[index] = 0;
    }
    else {
        count[index] = 0;
    }
    start[index] = 0;
}

CUDA_CALLABLE_MEMBER keyType Tree::getParticleKey(Particles *particles, integer index, integer maxLevel) {

    int level = 0;
    keyType particleKey = (keyType)0;

    int sonBox = 0;
    float min_x = *minX;
    float max_x = *maxX;
    float min_y = *minY;
    float max_y = *maxY;
    float min_z = *minZ;
    float max_z = *maxZ;

    // calculate path to the particle's position assuming an octree with above bounding boxes
    while (level <= maxLevel) {
        sonBox = 0;
        // find insertion point for body
        if (particles->x[index] < 0.5 * (min_x+max_x)) {
            sonBox += 1;
            max_x = 0.5 * (min_x+max_x);
        }
        else { min_x = 0.5 * (min_x+max_x); }
#if DIM > 1
        if (particles->y[index] < 0.5 * (min_y+max_y)) {
            sonBox += 2;
            max_y = 0.5 * (min_y + max_y);
        }
        else { min_y = 0.5 * (min_y + max_y); }
#if DIM == 3
        if (particles->z[index] < 0.5 * (min_z+max_z)) {
            sonBox += 4;
            max_z = 0.5 * (min_z + max_z);
        }
        else { min_z =  0.5 * (min_z + max_z); }
#endif
#endif
        particleKey = particleKey | ((unsigned long)sonBox << (unsigned long)(DIM * (maxLevel-level-1)));
        level ++;
    }
    //TODO: Hilbert change
    return particleKey;
    //return Lebesgue2Hilbert(particleKey, 21);
}

CUDA_CALLABLE_MEMBER Tree::~Tree() {

}

__global__ void TreeNS::Kernel::computeBoundingBox(Tree *tree, Particles *particles, integer *mutex, integer n,
                                                 integer blockSize) {

    integer index = threadIdx.x + blockDim.x * blockIdx.x;
    integer stride = blockDim.x * gridDim.x;

    // initialize local min/max
    //if (particles->x[index] != 0.f || particles->y[index] != 0.f || particles->z[index] != 0.f) {
    //    printf("device: x = (%f, %f, %f)\n", particles->x[index], particles->y[index], particles->z[index]);
    //}

    real x_min = particles->x[index];
    real x_max = particles->x[index];
#if DIM > 1
    real y_min = particles->y[index];
    real y_max = particles->y[index];
#if DIM == 3
    real z_min = particles->z[index];
    real z_max = particles->z[index];
#endif
#endif

    //if (index % 1000 == 0) {
    //    printf("device: x_min[%i] = %f\n", index, x_min);
    //}

    extern __shared__ real buffer[];

    real* x_min_buffer = (real*)buffer;
    real* x_max_buffer = (real*)&x_min_buffer[blockSize];
#if DIM > 1
    real* y_min_buffer = (real*)&x_max_buffer[blockSize];
    real* y_max_buffer = (real*)&y_min_buffer[blockSize];
#if DIM == 3
    real* z_min_buffer = (real*)&y_max_buffer[blockSize];
    real* z_max_buffer = (real*)&z_min_buffer[blockSize];
#endif
#endif

    integer offset = stride;

    // find (local) min/max
    while (index + offset < n) {

        x_min = fminf(x_min, particles->x[index + offset]);
        x_max = fmaxf(x_max, particles->x[index + offset]);
#if DIM > 1
        y_min = fminf(y_min, particles->y[index + offset]);
        y_max = fmaxf(y_max, particles->y[index + offset]);
#if DIM == 3
        z_min = fminf(z_min, particles->z[index + offset]);
        z_max = fmaxf(z_max, particles->z[index + offset]);
#endif
#endif

        offset += stride;
    }

    // save value in corresponding buffer
    x_min_buffer[threadIdx.x] = x_min;
    x_max_buffer[threadIdx.x] = x_max;
#if DIM > 1
    y_min_buffer[threadIdx.x] = y_min;
    y_max_buffer[threadIdx.x] = y_max;
#if DIM == 3
    z_min_buffer[threadIdx.x] = z_min;
    z_max_buffer[threadIdx.x] = z_max;
#endif
#endif

    // synchronize threads / wait for unfinished threads
    __syncthreads();

    integer i = blockDim.x/2; // assuming blockDim.x is a power of 2!

    // reduction within block
    while (i != 0) {
        if (threadIdx.x < i) {
            x_min_buffer[threadIdx.x] = fminf(x_min_buffer[threadIdx.x], x_min_buffer[threadIdx.x + i]);
            x_max_buffer[threadIdx.x] = fmaxf(x_max_buffer[threadIdx.x], x_max_buffer[threadIdx.x + i]);
#if DIM > 1
            y_min_buffer[threadIdx.x] = fminf(y_min_buffer[threadIdx.x], y_min_buffer[threadIdx.x + i]);
            y_max_buffer[threadIdx.x] = fmaxf(y_max_buffer[threadIdx.x], y_max_buffer[threadIdx.x + i]);
#if DIM == 3
            z_min_buffer[threadIdx.x] = fminf(z_min_buffer[threadIdx.x], z_min_buffer[threadIdx.x + i]);
            z_max_buffer[threadIdx.x] = fmaxf(z_max_buffer[threadIdx.x], z_max_buffer[threadIdx.x + i]);
#endif
#endif
        }
        __syncthreads();
        i /= 2;
    }

    // combining the results and generate the root cell
    if (threadIdx.x == 0) {
        while (atomicCAS(mutex, 0 ,1) != 0); // lock

        *tree->minX = fminf(*tree->minX, x_min_buffer[0]);
        *tree->maxX = fmaxf(*tree->maxX, x_max_buffer[0]);
#if DIM > 1
        *tree->minY = fminf(*tree->minY, y_min_buffer[0]);
        *tree->maxY = fmaxf(*tree->maxY, y_max_buffer[0]);
#if DIM == 3
        *tree->minZ = fminf(*tree->minZ, z_min_buffer[0]);
        *tree->maxZ = fmaxf(*tree->maxZ, z_max_buffer[0]);
#endif
#endif

        //if (*tree->minX != 0.f || *tree->minX) {
        //    printf("device: min/max: x = (%f, %f), y = (%f, %f), z = (%f, %f)\n", *tree->minX, *tree->maxX,
        //           *tree->minY, *tree->maxY, *tree->minZ, *tree->maxZ);
        //}

        atomicExch(mutex, 0); // unlock
    }
}

__global__ void TreeNS::Kernel::buildTree(Tree *tree, Particles *particles, integer n, integer m) {

    integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
    integer stride = blockDim.x * gridDim.x;

    //note: -1 used as "null pointer"
    //note: -2 used to lock a child (pointer)

    integer offset;
    bool newBody = true;

    real min_x;
    real max_x;
#if DIM > 1
    real min_y;
    real max_y;
#if DIM == 3
    real min_z;
    real max_z;
#endif
#endif

    integer childPath;
    integer temp;
    integer tempTemp;

    offset = 0;

    while ((bodyIndex + offset) < n) {

        if (newBody) {

            newBody = false;

            // copy bounding box
            min_x = *tree->minX;
            max_x = *tree->maxX;
#if DIM > 1
            min_y = *tree->minY;
            max_y = *tree->maxY;
#if DIM == 3
            min_z = *tree->minZ;
            max_z = *tree->maxZ;
#endif
#endif
            temp = 0;
            childPath = 0;

            // find insertion point for body
            if (particles->x[bodyIndex + offset] < 0.5 * (min_x + max_x)) { // x direction
                childPath += 1;
                max_x = 0.5 * (min_x + max_x);
            }
            else {
                min_x = 0.5 * (min_x + max_x);
            }
#if DIM > 1
            if (particles->y[bodyIndex + offset] < 0.5 * (min_y + max_y)) { // y direction
                childPath += 2;
                max_y = 0.5 * (min_y + max_y);
            }
            else {
                min_y = 0.5 * (min_y + max_y);
            }
#if DIM == 3
            if (particles->z[bodyIndex + offset] < 0.5 * (min_z + max_z)) {  // z direction
                childPath += 4;
                max_z = 0.5 * (min_z + max_z);
            }
            else {
                min_z = 0.5 * (min_z + max_z);
            }
#endif
#endif
        }

        integer childIndex = tree->child[temp*POW_DIM + childPath];

        // traverse tree until hitting leaf node
        while (childIndex >= m) { //n

            tempTemp = temp;
            temp = childIndex;

            childPath = 0;

            // find insertion point for body
            if (particles->x[bodyIndex + offset] < 0.5 * (min_x + max_x)) { // x direction
                childPath += 1;
                max_x = 0.5 * (min_x + max_x);
            }
            else {
                min_x = 0.5 * (min_x + max_x);
            }
#if DIM > 1
            if (particles->y[bodyIndex + offset] < 0.5 * (min_y + max_y)) { // y direction
                childPath += 2;
                max_y = 0.5 * (min_y + max_y);
            }
            else {
                min_y = 0.5 * (min_y + max_y);
            }
#if DIM == 3
            if (particles->z[bodyIndex + offset] < 0.5 * (min_z + max_z)) { // z direction
                childPath += 4;
                max_z = 0.5 * (min_z + max_z);
            }
            else {
                min_z = 0.5 * (min_z + max_z);
            }
#endif
#endif
            if (particles->mass[bodyIndex + offset] != 0) {
                atomicAdd(&particles->x[temp], particles->mass[bodyIndex + offset] * particles->x[bodyIndex + offset]);
#if DIM > 1
                atomicAdd(&particles->y[temp], particles->mass[bodyIndex + offset] * particles->y[bodyIndex + offset]);
#if DIM == 3
                atomicAdd(&particles->z[temp], particles->mass[bodyIndex + offset] * particles->z[bodyIndex + offset]);
#endif
#endif
            }

            atomicAdd(&particles->mass[temp], particles->mass[bodyIndex + offset]);
            atomicAdd(&tree->count[temp], 1);

            childIndex = tree->child[8*temp + childPath];
        }

        // if child is not locked
        if (childIndex != -2) {

            integer locked = temp * 8 + childPath;

            if (atomicCAS(&tree->child[locked], childIndex, -2) == childIndex) {

                // check whether a body is already stored at the location
                if (childIndex == -1) {
                    //insert body and release lock
                    tree->child[locked] = bodyIndex + offset;
                }
                else {
                    if (childIndex >= n) {
                        printf("ATTENTION!\n");
                    }
                    integer patch = POW_DIM * m; //8*n
                    while (childIndex >= 0 && childIndex < n) { // was n

                        //create a new cell (by atomically requesting the next unused array index)
                        integer cell = atomicAdd(tree->index, 1);
                        patch = min(patch, cell);

                        if (patch != cell) {
                            tree->child[8 * temp + childPath] = cell;
                        }

                        // insert old/original particle
                        childPath = 0;
                        if (particles->x[childIndex] < 0.5 * (min_x + max_x)) { childPath += 1; }
#if DIM > 1
                        if (particles->y[childIndex] < 0.5 * (min_y + max_y)) { childPath += 2; }
#if DIM == 3
                        if (particles->z[childIndex] < 0.5 * (min_z + max_z)) { childPath += 4; }
#endif
#endif

                        particles->x[cell] += particles->mass[childIndex] * particles->x[childIndex];
#if DIM > 1
                        particles->y[cell] += particles->mass[childIndex] * particles->y[childIndex];
#if DIM == 3
                        particles->z[cell] += particles->mass[childIndex] * particles->z[childIndex];
#endif
#endif

                        particles->mass[cell] += particles->mass[childIndex];
                        tree->count[cell] += tree->count[childIndex];

                        tree->child[POW_DIM * cell + childPath] = childIndex;
                        tree->start[cell] = -1;

                        // insert new particle
                        tempTemp = temp;
                        temp = cell;
                        childPath = 0;

                        // find insertion point for body
                        if (particles->x[bodyIndex + offset] < 0.5 * (min_x + max_x)) {
                            childPath += 1;
                            max_x = 0.5 * (min_x + max_x);
                        } else {
                            min_x = 0.5 * (min_x + max_x);
                        }
#if DIM > 1
                        if (particles->y[bodyIndex + offset] < 0.5 * (min_y + max_y)) {
                            childPath += 2;
                            max_y = 0.5 * (min_y + max_y);
                        } else {
                            min_y = 0.5 * (min_y + max_y);
                        }
#if DIM == 3
                        if (particles->z[bodyIndex + offset] < 0.5 * (min_z + max_z)) {
                            childPath += 4;
                            max_z = 0.5 * (min_z + max_z);
                        } else {
                            min_z = 0.5 * (min_z + max_z);
                        }
#endif
#endif

                        // COM / preparing for calculation of COM
                        if (particles->mass[bodyIndex + offset] != 0) {
                            particles->x[cell] += particles->mass[bodyIndex + offset] * particles->x[bodyIndex + offset];
#if DIM > 1
                            particles->y[cell] += particles->mass[bodyIndex + offset] * particles->y[bodyIndex + offset];
#if DIM == 3
                            particles->z[cell] += particles->mass[bodyIndex + offset] * particles->z[bodyIndex + offset];
#endif
#endif
                            particles->mass[cell] += particles->mass[bodyIndex + offset];
                        }
                        tree->count[cell] += tree->count[bodyIndex + offset];
                        childIndex = tree->child[POW_DIM * temp + childPath];
                    }

                    tree->child[POW_DIM * temp + childPath] = bodyIndex + offset;

                    __threadfence();  // written to global memory arrays (child, x, y, mass) thus need to fence
                    tree->child[locked] = patch;
                }
                offset += stride;
                newBody = true;
            }
        }
        __syncthreads();
    }
}

__global__ void TreeNS::Kernel::centerOfMass(Tree *tree, Particles *particles, integer n) {

    integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
    integer stride = blockDim.x*gridDim.x;
    integer offset = 0;

    //note: most of it already done within buildTreeKernel
    bodyIndex += n;

    while (bodyIndex + offset < *tree->index) {

        //TODO: check for mass = 0
        //if (particles->mass[bodyIndex + offset] == 0) {
        //    printf("centreOfMassKernel: mass = 0 (%i)!\n", bodyIndex + offset);
        //}

        if (particles->mass != 0) {
            particles->x[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
#if DIM > 1
            particles->y[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
#if DIM == 3
            particles->z[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
#endif
#endif
        }

        offset += stride;
    }
}

__global__ void TreeNS::Kernel::sort(Tree *tree, integer n, integer m) {

    integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
    integer stride = blockDim.x * gridDim.x;
    integer offset = 0;

    if (bodyIndex == 0) {
        integer sumParticles = 0;
        for (integer i=0; i<POW_DIM; i++) {
            sumParticles += tree->count[tree->child[i]];
        }
        printf("sumParticles = %i\n", sumParticles);
    }

    integer s = 0;
    if (threadIdx.x == 0) {

        for (integer i=0; i<POW_DIM; i++){

            integer node = tree->child[i];
            // not a leaf node
            if (node >= m) { //n
                tree->start[node] = s;
                s += tree->count[node];
            }
                // leaf node
            else if (node >= 0) {
                tree->sorted[s] = node;
                s++;
            }
        }
    }
    integer cell = m + bodyIndex;
    integer ind = *tree->index;

    //integer counter = 0; // for debugging purposes or rather to achieve the kernel to be finished
    while ((cell + offset) < ind /*&& counter < 100000*/) {
        //counter++;

        //if (counter > 99998) {
        //printf("cell + offset = %i\n", cell+offset);
        //}

        s = tree->start[cell + offset];

        if (s >= 0) {

            for (integer i=0; i<8; i++) {
                integer node = tree->child[POW_DIM*(cell+offset) + i];
                // not a leaf node
                if (node >= m) { //m
                    tree->start[node] = s;
                    s += tree->count[node];
                }
                    // leaf node
                else if (node >= 0) {
                    tree->sorted[s] = node;
                    s++;
                }
            }
            offset += stride;
        }
    }
}

namespace TreeNS {

    namespace Kernel {

        __global__ void set(Tree *tree, integer *count, integer *start, integer *child, integer *sorted,
                                  integer *index, real *minX, real *maxX) {
            tree->set(count, start, child, sorted, index, minX, maxX);
        }

        void Launch::set(Tree *tree, integer *count, integer *start, integer *child, integer *sorted,
                             integer *index, real *minX, real *maxX) {
            ExecutionPolicy executionPolicy(1, 1);
            cuda::launch(false, executionPolicy, ::TreeNS::Kernel::set, tree, count, start, child, sorted,
                         index, minX, maxX);
        }

#if DIM > 1

        __global__ void set(Tree *tree, integer *count, integer *start, integer *child, integer *sorted,
                                  integer *index, real *minX, real *maxX, real *minY, real *maxY) {
            tree->set(count, start, child, sorted, index, minX, maxX, minY, maxY);
        }

        void Launch::set(Tree *tree, integer *count, integer *start, integer *child, integer *sorted,
                             integer *index, real *minX, real *maxX, real *minY, real *maxY) {
            ExecutionPolicy executionPolicy(1, 1);
            cuda::launch(false, executionPolicy, ::TreeNS::Kernel::set, tree, count, start, child, sorted, index,
                         minX, maxX, minY, maxY);
        }

#if DIM == 3

        __global__ void set(Tree *tree, integer *count, integer *start, integer *child, integer *sorted,
                                  integer *index, real *minX, real *maxX, real *minY, real *maxY,
                                  real *minZ, real *maxZ) {
            tree->set(count, start, child, sorted, index, minX, maxX, minY, maxY, minZ, maxZ);
        }

        void Launch::set(Tree *tree, integer *count, integer *start, integer *child, integer *sorted,
                             integer *index, real *minX, real *maxX, real *minY, real *maxY,
                             real *minZ, real *maxZ) {
            ExecutionPolicy executionPolicy(1, 1);
            cuda::launch(false, executionPolicy, ::TreeNS::Kernel::set, tree, count, start, child, sorted, index, minX,
                         maxX, minY, maxY, minZ, maxZ);
        }

#endif
#endif


        __global__ void getParticleKeys(Tree *tree, Particles *particles, keyType *keys, integer maxLevel,
                                              integer n) {

            int bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            int stride = blockDim.x * gridDim.x;
            int offset = 0;

            unsigned long particleKey;
            unsigned long hilbertParticleKey;


            while (bodyIndex + offset < n) {

                //particleKey = 0UL;
                particleKey = tree->getParticleKey(particles, bodyIndex + offset, maxLevel);

                //TODO: Hilbert key
                //hilbertParticleKey = Lebesgue2Hilbert(particleKey, 21);
                keys[bodyIndex + offset] = particleKey; //hilbertParticleKey;

                offset += stride;
            }
        }

        namespace Launch {
            real buildTree(Tree *tree, Particles *particles, integer n, integer m, bool time) {
                ExecutionPolicy executionPolicy;
                return cuda::launch(time, executionPolicy, ::TreeNS::Kernel::buildTree, tree, particles, n, m);
            }

            real computeBoundingBox(Tree *tree, Particles *particles, integer *mutex, integer n, integer blockSize,
                                    bool time) {
                size_t sharedMemory = 6 * sizeof(real) * blockSize;
                //ExecutionPolicy executionPolicy(1024, 256, sharedMemory);
                ExecutionPolicy executionPolicy(256, 256, sharedMemory);
                return cuda::launch(time, executionPolicy, ::TreeNS::Kernel::computeBoundingBox, tree, particles, mutex, n, blockSize);
            }

            real centerOfMass(Tree *tree, Particles *particles, integer n, bool time) {
                ExecutionPolicy executionPolicy;
                return cuda::launch(time, executionPolicy, ::TreeNS::Kernel::centerOfMass, tree, particles, n);
            }

            real sort(Tree *tree, integer n, integer m, bool time) {
                ExecutionPolicy executionPolicy;
                return cuda::launch(time, executionPolicy, ::TreeNS::Kernel::sort, tree, n, m);
            }

            real getParticleKeys(Tree *tree, Particles *particles, keyType *keys, integer maxLevel, integer n,
                                 bool time) {
                ExecutionPolicy executionPolicy;
                return cuda::launch(time, executionPolicy, ::TreeNS::Kernel::getParticleKeys, tree, particles, keys, maxLevel, n);
            }
        }
    }
}
