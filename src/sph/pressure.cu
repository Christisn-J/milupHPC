#include "hip/hip_runtime.h"
#include "../../include/sph/pressure.cuh"
#include "../include/cuda_utils/cuda_launcher.cuh"


namespace EOS {
    __device__ void polytropicGas(Material *materials, Particles *particles, int index) {
        //printf("polytropicGas...\n");
        particles->p[index] = materials[particles->materialId[index]].eos.polytropic_K *
                pow(particles->rho[index], materials[particles->materialId[index]].eos.polytropic_gamma);
        //if (true /*particles->p[index] > 0.*/) {
        //    printf("pressure: p[%i] = %f, rho[%i] = %f, polyTropic_K = %f, polytropic_gamma = %f\n", index,
        //           particles->p[index], index, particles->rho[index], materials[particles->materialId[index]].eos.polytropic_K,
        //           materials[particles->materialId[index]].eos.polytropic_K);
        //}
    }

    __device__ void murnaghan(Material *materials, Particles *particles, int index){
        particles->p[index]= (materials[particles->materialId[index]].eos.bulk_modulus/materials[particles->materialId[index]].eos.n)*
                (pow(particles->rho[index]/materials[particles->materialId[index]].eos.rho_0, materials[particles->materialId[index]].eos.n) - 1.0);
    }

	__device__ void tillotson(Material *materials, Particles *particles, int index) {
    // Check if the energy is within the compressed region (e <= E_iv)
    if (particles->e[index] <= materials[particles->materialId[index]].eos.till_E_iv) {
        // Compressed region EOS
        particles->p[index] = 
            (materials[particles->materialId[index]].eos.till_a
			+ materials[particles->materialId[index]].eos.till_b / (1 + particles->e[index] / (materials[particles->materialId[index]].eos.till_E_0 * pow((particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0), 2))))
            * (particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0) * particles->e[index]
            + materials[particles->materialId[index]].eos.till_A * (particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0)
            + materials[particles->materialId[index]].eos.till_B * pow((particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0),2);
    	}
    // Check if the energy is within the expanded region (e >= E_cv)
    else if (particles->e[index] >= materials[particles->materialId[index]].eos.till_E_cv) {
        // Expanded region EOS
        particles->p[index] =
			materials[particles->materialId[index]].eos.till_a
			* (particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0) * particles->e[index]
			+ (materials[particles->materialId[index]].eos.till_b / (1 + particles->e[index] / (materials[particles->materialId[index]].eos.till_E_0 * pow((particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0), 2)))
			* (particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0) * particles->e[index]
			+ materials[particles->materialId[index]].eos.till_A * (particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0)
			* exp(-materials[particles->materialId[index]].eos.till_beta * ((materials[particles->materialId[index]].eos.till_rho_0 / particles->rho[index]) - 1)))
            * exp(-materials[particles->materialId[index]].eos.till_alpha * pow(((materials[particles->materialId[index]].eos.till_rho_0 / particles->rho[index]) - 1), 2));
    	}
    // Partial vaporization region (E_iv < e < E_cv)
    else {
        // Interpolate between the two regions (compressed and expanded)
        particles->p[index] = (1 -
			// interpolation weight
			(particles->e[index] - materials[particles->materialId[index]].eos.till_E_iv) / (materials[particles->materialId[index]].eos.till_E_cv - materials[particles->materialId[index]].eos.till_E_iv)) *
			//Compressed region EOS
            (
                (materials[particles->materialId[index]].eos.till_a
                + materials[particles->materialId[index]].eos.till_b / (1 + particles->e[index] / (materials[particles->materialId[index]].eos.till_E_0 * pow((particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0), 2))))
                * (particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0) * particles->e[index]
                + materials[particles->materialId[index]].eos.till_A * (particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0)
                + materials[particles->materialId[index]].eos.till_B * pow((particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0),2)
            ) +
			// interpolation weight
            (particles->e[index] - materials[particles->materialId[index]].eos.till_E_iv) / (materials[particles->materialId[index]].eos.till_E_cv - materials[particles->materialId[index]].eos.till_E_iv) *
			//Expanded region EOS
            (
                materials[particles->materialId[index]].eos.till_a
                * (particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0) * particles->e[index]
                + (materials[particles->materialId[index]].eos.till_b / (1 + particles->e[index] / (materials[particles->materialId[index]].eos.till_E_0 * pow((particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0), 2)))
                * (particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0) * particles->e[index]
                + materials[particles->materialId[index]].eos.till_A * (particles->rho[index] / materials[particles->materialId[index]].eos.till_rho_0)
                * exp(-materials[particles->materialId[index]].eos.till_beta * ((materials[particles->materialId[index]].eos.till_rho_0 / particles->rho[index]) - 1)))
                * exp(-materials[particles->materialId[index]].eos.till_alpha * pow(((materials[particles->materialId[index]].eos.till_rho_0 / particles->rho[index]) - 1), 2))
            );
    	}
	}

    __device__ void isothermalGas(Material *materials, Particles *particles, int index) {
        //printf("isothermalGas...\n");
        particles->p[index] = 41255.407 * particles->rho[index];
    }

    __device__ void idealGas(Material *materials, Particles *particles, int index) {
        //printf("idealGas...\n");
        //if (index % 1000 == 0) {
        //    printf("polytropic gamma: %e\n", materials[particles->materialId[index]].eos.polytropic_gamma);
        //}
        particles->p[index] = (materials[particles->materialId[index]].eos.polytropic_gamma - 1) *
                        particles->rho[index] * particles->e[index];
        if (particles->p[index] < 0) {
            printf("negative pressure! p[%i] = %e, rho = %e, e = %e\n", index, particles->p[index], particles->rho[index], particles->e[index]);
        }
        //particles->p[index] = particles->cs[index] * particles->cs[index] * particles->rho[index];
    }

    __device__ void locallyIsothermalGas(Material *materials, Particles *particles, int index) {
        //printf("locallyIsothermalGas...\n");
        particles->p[index] = particles->cs[index] * particles->cs[index] * particles->rho[index];
    }
}

namespace SPH {
    namespace Kernel {
        __global__ void calculatePressure(Material *materials, Particles *particles, int numParticles) {

            register int i, inc;
            register double eta, e, rho, mu, p1, p2;
            int i_rho, i_e;
            double pressure;

            inc = blockDim.x * gridDim.x;
            for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {

                pressure = 0.0;

                //printf("calculatePressure: %i\n", materials[particles->materialId[i]].eos.type);
                switch (materials[particles->materialId[i]].eos.type) {
                    case EquationOfStates::EOS_TYPE_POLYTROPIC_GAS: {
                        ::EOS::polytropicGas(materials, particles, i);
                    }
                        break;
                    case EquationOfStates::EOS_TYPE_MURNAGHAN: {
                        ::EOS::murnaghan(materials, particles, i);
                    }
                        break;
					case EquationOfStates::EOS_TYPE_TILLOTSON: {
                        ::EOS::tillotson(materials, particles, i);
                    }
                        break;
                    case EquationOfStates::EOS_TYPE_ISOTHERMAL_GAS: {
                        ::EOS::isothermalGas(materials, particles, i);
                    }
                        break;
                    case EquationOfStates::EOS_TYPE_IDEAL_GAS: {
                        ::EOS::idealGas(materials, particles, i);
                    }
                        break;
                    case EquationOfStates::EOS_TYPE_LOCALLY_ISOTHERMAL_GAS: {
                        ::EOS::locallyIsothermalGas(materials, particles, i);
                    }
                        break;
                    default:
                        printf("not implemented!\n");
                }

            }
        }

        real Launch::calculatePressure(Material *materials, Particles *particles, int numParticles) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::SPH::Kernel::calculatePressure, materials,
                                particles, numParticles);
        }
    }
}

