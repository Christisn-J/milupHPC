#include "hip/hip_runtime.h"
#include "../../include/sph/density.cuh"
#include "../include/cuda_utils/cuda_launcher.cuh"

namespace SPH {

    namespace Kernel {

        __global__ void calculateDensity(::SPH::SPH_kernel kernel, Particles *particles, int *interactions, int numParticles) {

            int i;
            int j;
            int inc;
            int ip;
            int d;
            real W;
            real Wj;
            real dx[DIM];
            real dWdx[DIM];
            real dWdr;
            real rho;
            real sml;
            real tolerance;
//#if SML_CORRECTION
//    double dhdrho, sml_omega,sml_omega_sum, r;
//    double f, df, h_new, h_init, rho_h;
//    //the proportionality constant (h_fact = 4.0) defines the average number of neighbours: [2D] noi = pi * h_fact^2, [3D] noi = 4/3 * pi * h_fact^3
//    double h_fact = 4.0;
//#endif // SML_CORRECTION

            inc = blockDim.x * gridDim.x;
            for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i += inc) {
                //if (EOS_TYPE_IGNORE == matEOS[p_rhs.materialId[i]] || p_rhs.materialId[i] == EOS_TYPE_IGNORE) {
                //    continue;
                //}
                tolerance = 0.0;
                int cnt = 0;

//#if SML_CORRECTION
//        h_init = p.h[i];
//        h_new = 0.0;
//        /* // if Bisection method is used
//        double a = 0.0, b = 0.0, c = 0.0;
//	    int bis_cnt = 0;
//        int bisection = 0; */
//#endif // SML_CORRECTION

                do {
//#if SML_CORRECTION
//            sml_omega_sum = 0.0;
//#endif // SML_CORRECTION
                    sml = particles->sml[i];

                    // self density is m_i W_ii
                    for (d = 0; d < DIM; d++) {
                        dx[d] = 0;
                    }

                    kernel(&W, dWdx, &dWdr, dx, sml);
//#if SHEPARD_CORRECTION
//            W /= p_rhs.shepard_correction[i];
//#endif
                    rho = particles->mass[i] * W;
                    //if (rho == 0.0) {
                    //    printf("rho is %f W: %e \n", rho, W);
                    //}
                    // sph sum for particle i
                    for (j = 0; j < particles->noi[i]; j++) {
                        ip = interactions[i * MAX_NUM_INTERACTIONS + j];
                        //if (EOS_TYPE_IGNORE == matEOS[p_rhs.materialId[ip]] || p_rhs.materialId[ip] == EOS_TYPE_IGNORE) {
                        //    continue;
                        //}
//#if (VARIABLE_SML || INTEGRATE_SML || DEAL_WITH_TOO_MANY_INTERACTIONS)
//                sml = 0.5*(p.h[i] + p.h[ip]);
//#endif

                        dx[0] = particles->x[i] - particles->x[ip];
#if DIM > 1
                        dx[1] = particles->y[i] - particles->y[ip];
#if DIM > 2
                        dx[2] = particles->z[i] - particles->z[ip];
#endif
#endif

//#if SML_CORRECTION
//                r = 0;
//                for (d = 0; d < DIM; d++) {
//                    r += dx[d]*dx[d];
//                }
//                r = sqrt(r);
//#endif // SML_CORRECTION

//#if AVERAGE_KERNELS
//                kernel(&W, dWdx, &dWdr, dx, p.h[i]);
//                Wj = 0;
//                kernel(&Wj, dWdx, &dWdr, dx, p.h[j]);
//# if SHEPARD_CORRECTION
//                W /= p_rhs.shepard_correction[i];
//                Wj /= p_rhs.shepard_correction[j];
//# endif
//                W = 0.5 * (W + Wj);
//#else
                        kernel(&W, dWdx, &dWdr, dx, sml);
//# if SHEPARD_CORRECTION
//                W /= p_rhs.shepard_correction[i];
//# endif
//                // contribution of interaction
//#endif // AVERAGE_KERNELS

//#if SML_CORRECTION
//                sml_omega_sum += p.m[ip] * (-1) * (DIM * W/sml + (r / sml) * dWdr);
//#endif // SML_CORRECTION
                        rho += particles->mass[ip] * W;
                    }
//#if SML_CORRECTION
//            rho_h = p.m[i] * pow(double(h_fact / p.h[i]), DIM);
//            dhdrho = -p.h[i] / (DIM * rho);
//            sml_omega = 1 - dhdrho * sml_omega_sum;
//
//            // Newton-Raphson method tolerance e-3 (Phantom)
//            f = rho_h - rho;
//            df = -DIM * rho / p.h[i] * sml_omega;
//            h_new = p.h[i] - f / df;
//
//            // arbitrary set limit for sml change
//            if (h_new > 1.2 * p.h[i]) {
//                h_new = 1.2 * p.h[i];
//            } else if (h_new < 0.8 * p.h[i]) {
//                h_new = 0.8 * p.h[i];
//            }
//
//           	tolerance = abs(h_new - p.h[i]) / h_init;
//            if (tolerance > 1e-3) {
//                if (h_new < 0){
//	       	        printf("SML_CORRECTION: NEGATIVE SML!");
//                }
//                p.h[i] = h_new;
//                p.sml_omega[i] = sml_omega;
//                redo_NeighbourSearch(i, interactions);
//                cnt++;
//            }
//#endif // SML_CORRECTION

                } while (tolerance > 1e-3 && cnt < 10);
                // write to global memory
                particles->rho[i] = rho;
                //if (particles->rho[i] > 0.) {
                //    printf("density: rho[%i] = %f\n", i, particles->rho[i]);
                //}
            }
        }

        real Launch::calculateDensity(::SPH::SPH_kernel kernel, Particles *particles, int *interactions, int numParticles) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::SPH::Kernel::calculateDensity, kernel, particles, interactions, numParticles);
        }

    }
}
