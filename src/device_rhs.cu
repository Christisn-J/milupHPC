#include "hip/hip_runtime.h"
#include "../include/device_rhs.cuh"
#include "../include/cuda_utils/cuda_launcher.cuh"

#if TARGET_GPU
namespace Kernel {

    __global__ void resetArrays(Tree *tree, Particles *particles, integer *mutex, integer n, integer m) {

        integer bodyIndex = threadIdx.x + blockDim.x*blockIdx.x;
        integer stride = blockDim.x*gridDim.x;
        integer offset = 0;

        while ((bodyIndex + offset) < m) {
            tree->reset(bodyIndex + offset, n);

            if ((bodyIndex + offset) >= n) {
                particles->reset(bodyIndex + offset);
            }

            offset += stride;
        }

        if (bodyIndex == 0) {
            *mutex = 0;
            *tree->index = n;
            *tree->minX = 0;
            *tree->maxX = 0;
#if DIM > 1
            *tree->minY = 0;
            *tree->maxY = 0;
#if DIM == 3
            *tree->minZ = 0;
            *tree->maxZ = 0;
#endif
#endif
            tree->toDeleteLeaf[0] = -1;
            tree->toDeleteLeaf[1] = -1;
            tree->toDeleteNode[0] = -1;
            tree->toDeleteNode[1] = -1;

        }
    }

    real Launch::resetArrays(Tree *tree, Particles *particles, integer *mutex, integer n, integer m, bool time) {
        ExecutionPolicy executionPolicy;
        return cuda::launch(time, executionPolicy, ::Kernel::resetArrays, tree, particles, mutex,  n, m);
    }

}
#endif

