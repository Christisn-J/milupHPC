#include "hip/hip_runtime.h"
#include "../../include/gravity/gravity.cuh"
#include "../../include/cuda_utils/cuda_launcher.cuh"

namespace Gravity {

    namespace Kernel {

        __global__ void zeroDomainListNodes(Particles *particles, DomainList *domainList,
                                            DomainList *lowestDomainList) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;
            integer domainIndex;
            bool zero;

            while ((bodyIndex + offset) < *domainList->domainListIndex) {
                zero = true;
                domainIndex = domainList->domainListIndices[bodyIndex + offset];
                for (int i=0; i<*lowestDomainList->domainListIndex-1; i++) {
                    if (domainIndex = lowestDomainList->domainListIndices[i]) {
                        zero = false;
                    }
                }

                if (zero) {
                    particles->x[domainIndex] = 0.f;
                    particles->y[domainIndex] = 0.f;
                    particles->z[domainIndex] = 0.f;

                    particles->mass[domainIndex] = 0.f;
                }

                offset += stride;
            }

        }

        __global__ void prepareLowestDomainExchange(Particles *particles, DomainList *lowestDomainList,
                                                    Helper *helper, Entry::Name entry) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;
            integer index;
            integer lowestDomainIndex;

            //copy x, y, z, mass of lowest domain list nodes into arrays
            //sorting using cub (not here)
            while ((bodyIndex + offset) < *lowestDomainList->domainListIndex) {
                lowestDomainIndex = lowestDomainList->domainListIndices[bodyIndex + offset];
                if (lowestDomainIndex >= 0) {
                    switch (entry) {
                        case Entry::x:
                            helper->realBuffer[bodyIndex + offset] = particles->x[lowestDomainIndex];
                            //printf("lowestDomainIndex = %i: realBuffer = %f, x = %f\n", lowestDomainIndex,
                            //       helper->realBuffer[bodyIndex + offset], particles->x[lowestDomainIndex]);
                            break;
#if DIM > 1
                        case Entry::y:
                            helper->realBuffer[bodyIndex + offset] = particles->y[lowestDomainIndex];
                            break;
#if DIM == 3
                        case Entry::z:
                            helper->realBuffer[bodyIndex + offset] = particles->z[lowestDomainIndex];
                            break;
#endif
#endif
                        case Entry::mass:
                            helper->realBuffer[bodyIndex + offset] = particles->mass[lowestDomainIndex];
                            //printf("lowestDomainIndex = %i: realBuffer = %f, mass = %f\n", lowestDomainIndex,
                            //       helper->realBuffer[bodyIndex + offset], particles->mass[lowestDomainIndex]);
                            break;
                        default:
                            helper->realBuffer[bodyIndex + offset] = particles->mass[lowestDomainIndex];
                            break;
                    }
                }
                offset += stride;
            }
        }

        __global__ void updateLowestDomainListNodes(Particles *particles, DomainList *lowestDomainList,
                                                    Helper *helper, Entry::Name entry) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            integer originalIndex = -1;

            while ((bodyIndex + offset) < *lowestDomainList->domainListIndex) {
                for (int i = 0; i < *lowestDomainList->domainListIndex; i++) {
                    if (lowestDomainList->sortedDomainListKeys[bodyIndex + offset] ==
                        lowestDomainList->domainListKeys[i]) {
                        originalIndex = i;
                    }
                }

                if (originalIndex == -1) {
                    printf("ATTENTION: originalIndex = -1 (index = %i)!\n",
                           lowestDomainList->sortedDomainListKeys[bodyIndex + offset]);
                }

                switch (entry) {
                    case Entry::x:
                        particles->x[lowestDomainList->domainListIndices[originalIndex]] =
                                helper->realBuffer[DOMAIN_LIST_SIZE + bodyIndex + offset];
                        //printf("lowestDomainIndex: lowestDomainList->domainListIndices[%i] = %i == %f\n", originalIndex,
                        //       lowestDomainList->domainListIndices[originalIndex], helper->realBuffer[DOMAIN_LIST_SIZE + bodyIndex + offset]);
                        break;
#if DIM > 1
                    case Entry::y:
                        particles->y[lowestDomainList->domainListIndices[originalIndex]] =
                                helper->realBuffer[DOMAIN_LIST_SIZE + bodyIndex + offset];
                        break;
#if DIM == 3
                    case Entry::z:
                        particles->z[lowestDomainList->domainListIndices[originalIndex]] =
                                helper->realBuffer[DOMAIN_LIST_SIZE + bodyIndex + offset];
                        break;
#endif
#endif
                    case Entry::mass:
                        particles->mass[lowestDomainList->domainListIndices[originalIndex]] =
                                helper->realBuffer[DOMAIN_LIST_SIZE + bodyIndex + offset];
                        break;
                    default:
                        printf("Entry not available!\n");
                        break;
                }

                offset += stride;
            }
        }

        __global__ void compLowestDomainListNodes(Particles *particles, DomainList *lowestDomainList) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;
            integer lowestDomainIndex;

            while ((bodyIndex + offset) < *lowestDomainList->domainListIndex) {

                lowestDomainIndex = lowestDomainList->domainListIndices[bodyIndex + offset];

                if (particles->mass[lowestDomainIndex] != 0) {
                    particles->x[lowestDomainIndex] /= particles->mass[lowestDomainIndex];
#if DIM > 1
                    particles->y[lowestDomainIndex] /= particles->mass[lowestDomainIndex];
#if DIM == 3
                    particles->z[lowestDomainIndex] /= particles->mass[lowestDomainIndex];
#endif
#endif
                }
                offset += stride;
            }
        }

        __global__ void compLocalPseudoParticles(Tree *tree, Particles *particles, DomainList *domainList, int n) {
            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;
            bool isDomainList;
            //note: most of it already done within buildTreeKernel

            bodyIndex += n;

            while (bodyIndex + offset < *tree->index) {
                isDomainList = false;

                for (integer i=0; i<*domainList->domainListIndex; i++) {
                    if ((bodyIndex + offset) == domainList->domainListIndices[i]) {
                        isDomainList = true; // hence do not insert
                        break;
                    }
                }

                if (particles->mass[bodyIndex + offset] != 0 && !isDomainList) {
                    particles->x[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
#if DIM > 1
                    particles->y[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
#if DIM == 3
                    particles->z[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
#endif
#endif
                }

                offset += stride;
            }
        }

        __global__ void compDomainListPseudoParticles(Tree *tree, Particles *particles, DomainList *domainList,
                                                      DomainList *lowestDomainList, int n) {
            //calculate position (center of mass) and mass for domain list nodes
            //Problem: start with "deepest" nodes
            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset;

            integer domainIndex;
            integer level = MAX_LEVEL; // max level
            bool compute;

            // go from max level to level=0
            while (level >= 0) {
                offset = 0;
                compute = true;
                while ((bodyIndex + offset) < *domainList->domainListIndex) {
                    compute = true;
                    domainIndex = domainList->domainListIndices[bodyIndex + offset];
                    for (int i=0; i<*lowestDomainList->domainListIndex; i++) {
                        if (domainIndex == lowestDomainList->domainListIndices[i]) {
                            compute = false;
                        }
                    }
                    if (compute && domainList->domainListLevels[bodyIndex + offset] == level) {
                        // do the calculation
                        for (int i=0; i<POW_DIM; i++) {
                            particles->x[domainIndex] += particles->x[tree->child[POW_DIM*domainIndex + i]] *
                                    particles->mass[tree->child[POW_DIM*domainIndex + i]];
                            //printf("x += %f * %f = %f (%i)\n", particles->x[tree->child[POW_DIM*domainIndex + i]], particles->mass[tree->child[POW_DIM*domainIndex + i]],
                            //       particles->x[tree->child[POW_DIM*domainIndex + i]] * particles->mass[tree->child[POW_DIM*domainIndex + i]],
                            //       tree->child[POW_DIM*domainIndex + i]);
#if DIM > 1
                            particles->y[domainIndex] += particles->y[tree->child[POW_DIM*domainIndex + i]] *
                                    particles->mass[tree->child[POW_DIM*domainIndex + i]];
#if DIM == 3
                            particles->z[domainIndex] += particles->z[tree->child[POW_DIM*domainIndex + i]] *
                                    particles->mass[tree->child[POW_DIM*domainIndex + i]];
#endif
#endif
                            particles->mass[domainIndex] += particles->mass[tree->child[POW_DIM*domainIndex + i]];
                        }

                        if (particles->mass[domainIndex] != 0.f) {
                            particles->x[domainIndex] /= particles->mass[domainIndex];
#if DIM > 1
                            particles->y[domainIndex] /= particles->mass[domainIndex];
#if DIM == 3
                            particles->z[domainIndex] /= particles->mass[domainIndex];
#endif
#endif
                        }
                    }
                    offset += stride;
                }
                __syncthreads();
                level--;
            }
        }

        __global__ void computeForces(Tree *tree, Particles *particles, integer n, integer m, integer blockSize,
                                      integer warp, integer stackSize) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;

            //__shared__ float depth[stackSize * blockSize/warp];
            // stack controlled by one thread per warp
            //__shared__ int   stack[stackSize * blockSize/warp];
            extern __shared__ real buffer[];

            real* depth = (real*)buffer;
            real* stack = (real*)&depth[stackSize* blockSize/warp];

            real x_radius = 0.5*(*tree->maxX - (*tree->minX));
#if DIM > 1
            real y_radius = 0.5*(*tree->maxY - (*tree->minY));
#if DIM == 3
            real z_radius = 0.5*(*tree->maxZ - (*tree->minZ));
#endif
#endif

#if DIM == 1
            real radius = x_radius;
#elif DIM == 2
            real radius = fmaxf(x_radius, y_radius);
#else
            real radius_max = fmaxf(x_radius, y_radius);
            real radius = fmaxf(radius_max, z_radius);
#endif

            // in case that one of the first 8 children are a leaf
            integer jj = -1;
            for (integer i=0; i<POW_DIM; i++) {
                if (tree->child[i] != -1) {
                    jj++;
                }
            }

            integer counter = threadIdx.x % warp;
            integer stackStartIndex = stackSize*(threadIdx.x / warp);

            while ((bodyIndex + offset) < m) {

                integer sortedIndex = tree->sorted[bodyIndex + offset];

                //if ((bodyIndex + offset) % 1000 == 0) {
                //    printf("computeForces: sortedIndex = %i\n", sortedIndex);
                //}

                real pos_x = particles->x[sortedIndex];
#if DIM > 1
                real pos_y = particles->y[sortedIndex];
#if DIM == 3
                real pos_z = particles->z[sortedIndex];
#endif
#endif

                real acc_x = 0.0;
#if DIM > 1
                real acc_y = 0.0;
#if DIM == 3
                real acc_z = 0.0;
#endif
#endif

                // initialize stack
                integer top = jj + stackStartIndex;

                if (counter == 0) {

                    integer temp = 0;

                    for (int i=0; i<POW_DIM; i++) {
                        // if child is not locked
                        if (tree->child[i] != -1) {
                            stack[stackStartIndex + temp] = tree->child[i];
                            depth[stackStartIndex + temp] = radius*radius/theta;
                            temp++;
                        }
                    }
                }
                __syncthreads();

                // while stack is not empty / more nodes to visit
                while (top >= stackStartIndex) {

                    integer node = stack[top];
                    //debug
                    //if (node > n && node < m) {
                    //    printf("PARALLEL FORCE! (node = %i x = (%f, %f, %f) m = %f)\n", node, x[node], y[node], z[node],
                    //        mass[node]);
                    //}
                    //end: debug
                    real dp = 0.25*depth[top]; // float dp = depth[top];

                    for (integer i=0; i<POW_DIM; i++) {

                        integer ch = tree->child[POW_DIM*node + i];
                        //__threadfence();

                        if (ch >= 0) {

                            real dx = particles->x[ch] - pos_x;
#if DIM > 1
                            real dy = particles->y[ch] - pos_y;
#if DIM == 3
                            real dz = particles->z[ch] - pos_z;
#endif
#endif

                            real r = dx*dx + 0.05; //0.0025; //NEW: TODO: needed for smoothing
#if DIM > 1
                            r += dy*dy;
#if DIM == 3
                            r += dz*dz;
#endif
#endif

                            //unsigned activeMask = __activemask();

                            //if (ch < n /*is leaf node*/ || !__any_sync(activeMask, dp > r)) {
                            if (ch < n /*is leaf node*/ || __all_sync(__activemask(), dp <= r)) { //NEW: && ch != sortedIndex

                                /*//debug
                                key = getParticleKeyPerParticle(x[ch], y[ch], z[ch], minX, maxX, minY, maxY,
                                                                minZ, maxZ, 21);
                                if (key2proc(key, s) != s->rank) {
                                    printf("Parallel force! child = %i x = (%f, %f, %f) mass = %f\n", ch, x[ch], y[ch], z[ch], mass[ch]);
                                }
                                //end: debug*/

                                // calculate interaction force contribution
                                if (r > 0.f) { //NEW //TODO: how to avoid r = 0?
                                    r = rsqrt(r);
                                }
                                //if (r == 0.f) {
                                //    printf("r = 0!!! x[%i] = (%f, %f, %f) vs x[%i] = (%f, %f, %f)\n", sortedIndex,
                                //           particles->x[sortedIndex], particles->y[sortedIndex], particles->z[sortedIndex],
                                //           ch, particles->x[ch], particles->y[ch], particles->z[ch]);
                                //}
                                real f = particles->mass[ch] * r * r * r;// + 0.0025;



                                acc_x += f*dx; // * 0.0001;
#if DIM > 1
                                acc_y += f*dy; // * 0.0001;
#if DIM == 3
                                acc_z += f*dz; // * 0.0001;
#endif
#endif
                                /*if (acc_x > 500000) {
                                    printf("huge acceleration!!! r = %f acc = (%f, %f, %f) x[%i] = (%f, %f, %f) m = %f vs x[%i] = (%f, %f, %f) m = %f\n",
                                           r, acc_x, acc_y, acc_z, sortedIndex,
                                           particles->x[sortedIndex], particles->y[sortedIndex],
                                           particles->z[sortedIndex], particles->mass[sortedIndex],
                                           ch, particles->x[ch], particles->y[ch], particles->z[ch],
                                           particles->mass[ch]);
                                }*/
                                //if (particles->mass[ch] > 10000) {
                                //    printf("mass is huge for ch=%i with node = %i (mass = %f, r = %f, f = %f -> acc = (%f, %f, %f) f = (%f, %f, %f))!\n", ch, node, particles->mass[ch],
                                //    r, f, acc_x, acc_y, acc_z, f*dx, f*dy, f*dz);
                                //}
                            }
                            else {
                                // if first thread in warp: push node's children onto iteration stack
                                if (counter == 0) {
                                    stack[top] = ch;
                                    depth[top] = dp; // depth[top] = 0.25*dp;
                                }
                                top++; // descend to next tree level
                                //__threadfence();
                            }
                        }
                        else { /*top = max(stackStartIndex, top-1); */}
                    }
                    top--;
                }
                // update body data
                particles->ax[sortedIndex] = acc_x;
#if DIM > 1
                particles->ay[sortedIndex] = acc_y;
#if DIM == 3
                particles->az[sortedIndex] = acc_z;
#endif
#endif

                offset += stride;

                __syncthreads();
            }

        }

        __global__ void update(Particles *particles, integer n, real dt, real d) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            while (bodyIndex + offset < n) {

                /*if ((bodyIndex + offset) % 1000 == 0) {
                    printf("index= %i: velocity = = (%f, %f, %f)  acceleration = (%f, %f, %f)\n",
                           bodyIndex + offset,
                           particles->vx[bodyIndex + offset], particles->vy[bodyIndex + offset],
                           particles->vz[bodyIndex + offset],
                           particles->ax[bodyIndex + offset], particles->ay[bodyIndex + offset],
                           particles->az[bodyIndex + offset]);
                }*/

               // calculating/updating the velocities
                particles->vx[bodyIndex + offset] += dt * particles->ax[bodyIndex + offset];
#if DIM > 1
                particles->vy[bodyIndex + offset] += dt * particles->ay[bodyIndex + offset];
#if DIM == 3
                particles->vz[bodyIndex + offset] += dt * particles->az[bodyIndex + offset];
#endif
#endif

                // calculating/updating the positions
                particles->x[bodyIndex + offset] += d * dt * particles->vx[bodyIndex + offset];
#if DIM > 1
                particles->y[bodyIndex + offset] += d * dt * particles->vy[bodyIndex + offset];
#if DIM == 3
                particles->z[bodyIndex + offset] += d * dt * particles->vz[bodyIndex + offset];
#endif
#endif
                offset += stride;
            }
        }

        __global__ void createKeyHistRanges(Helper *helper, integer bins) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            keyType max_key = 1UL << 63;

            while ((bodyIndex + offset) < bins) {

                helper->keyTypeBuffer[bodyIndex + offset] = (bodyIndex + offset) * (max_key/bins);
                //printf("keyHistRanges[%i] = %lu\n", bodyIndex + offset, keyHistRanges[bodyIndex + offset]);

                if ((bodyIndex + offset) == (bins - 1)) {
                    helper->keyTypeBuffer[bins-1] = KEY_MAX;
                }
                offset += stride;
            }
        }

        __global__ void symbolicForce(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                      DomainList *domainList, Helper *helper,
                                      real diam, real theta_, integer n, integer m, integer relevantIndex,
                                      Curve::Type curveType) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;
            real r;
            integer insertIndex;
            bool insert;
            integer level;
            integer childIndex;
            //bool redo = false;

            while ((bodyIndex + offset) < *tree->index) {

                insert = true;
                //redo = false;

                for (integer i=0; i<*domainList->domainListIndex; i++) {
                    if ((bodyIndex + offset) == domainList->domainListIndices[i]) {
                        insert = false;
                        break;
                    }
                }

                //if (mass[relevantDomainListIndices[relevantIndex]] == 0) {
                //    insert = false;
                //}

                if (insert && (bodyIndex + offset) != domainList->relevantDomainListIndices[relevantIndex] &&
                    ((bodyIndex + offset) < subDomainKeyTree->procParticleCounter[subDomainKeyTree->rank] || (bodyIndex + offset) > n)) {

                    r = particles->distance(relevantIndex, bodyIndex + offset);
                    //r = smallestDistance(x, y, z, relevantDomainListIndices[relevantIndex], bodyIndex + offset);

                    //calculate tree level by determining the particle's key and traversing the tree until hitting that particle
                    level = tree->getTreeLevel(particles, bodyIndex + offset, MAX_LEVEL, curveType);
                    //level = getTreeLevel(bodyIndex + offset, child, x, y, z, minX, maxX, minY, maxY, minZ, maxZ);

                    if ((powf(0.5, level) * diam) >= (theta_ * r) && level >= 0) {
                        //TODO: insert cell itself or children?

                        /// inserting cell itself
                        /* //check whether node is a domain list node
                        for (int i=0; i<*domainList->domainListIndex; i++) {
                            if ((bodyIndex + offset) == domainList->domainListIndices[i]) {
                                insert = false;
                                break;
                                //printf("domain list nodes do not need to be sent!\n");

                            }
                        }
                        if (insert) {
                            //add to indices to be sent
                            insertIndex = atomicAdd(domainList->domainListCounter, 1);
                            //sendIndices[insertIndex] = bodyIndex + offset;
                            helper->integerBuffer[insertIndex] = bodyIndex + offset;
                        }
                        else {

                        }*/

                        /// inserting children
                        for (int i=0; i<POW_DIM; i++) {
                            childIndex = tree->child[POW_DIM * (bodyIndex + offset) + i];
                            //check whether node is already within the indices to be sent
                            //check whether node is a domain list node
                            for (int i = 0; i < *domainList->domainListIndex; i++) {
                                if (childIndex == domainList->domainListIndices[i]) {
                                    insert = false;
                                    //printf("domain list nodes do not need to be sent!\n");
                                }
                            }
                            if (insert && childIndex != -1) {
                                //add to indices to be sent
                                insertIndex = atomicAdd(domainList->domainListCounter, 1);
                                helper->integerBuffer[insertIndex] = childIndex;
                                //sendIndices[insertIndex] = childIndex;
                            }
                        }
                    }
                }
                else {
                    //no particle to examine...
                }
                offset += stride;
            }

        }

        __global__ void compTheta(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                  DomainList *domainList, Helper *helper, Curve::Type curveType) {

            integer index = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;
            integer bodyIndex;
            keyType key;
            integer domainIndex;

            //"loop" over domain list nodes
            while ((index + offset) < *domainList->domainListIndex) {

                bodyIndex = domainList->domainListIndices[index + offset];
                //calculate key
                key = tree->getParticleKey(particles, bodyIndex, MAX_LEVEL, curveType);
                //key = getParticleKeyPerParticle(x[bodyIndex], y[bodyIndex], z[bodyIndex], minX, maxX, minY, maxY,
                //                                minZ, maxZ, 21);

                //if domain list node belongs to other process: add to relevant domain list indices
                if (subDomainKeyTree->key2proc(key) != subDomainKeyTree->rank) {
                    domainIndex = atomicAdd(domainList->domainListCounter, 1);
                    domainList->relevantDomainListIndices[domainIndex] = bodyIndex;
                }
                offset += stride;
            }

        }

        __global__ void keyHistCounter(Tree *tree, Particles *particles, SubDomainKeyTree *subDomainKeyTree,
                                       Helper *helper,
                                       /*keyType *keyHistRanges, integer *keyHistCounts,*/ int bins, int n,
                                       Curve::Type curveType) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            keyType key;

            while ((bodyIndex + offset) < n) {

                key = tree->getParticleKey(particles, bodyIndex + offset, MAX_LEVEL, curveType);

                for (int i = 0; i < (bins); i++) {
                    if (key >= helper->keyTypeBuffer[i] && key < helper->keyTypeBuffer[i + 1]) {
                        //keyHistCounts[i] += 1;
                        atomicAdd(&helper->integerBuffer[i], 1);
                        break;
                    }
                }

                offset += stride;
            }

        }

        //TODO: resetting helper (buffers)?!
        __global__ void calculateNewRange(SubDomainKeyTree *subDomainKeyTree, Helper *helper,
                                          /*keyType *keyHistRanges, integer *keyHistCounts,*/ int bins, int n,
                                          Curve::Type curveType) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            integer sum;
            keyType newRange;

            while ((bodyIndex + offset) < (bins-1)) {

                sum = 0;
                for (integer i=0; i<(bodyIndex+offset); i++) {
                    sum += helper->integerBuffer[i];
                }

                for (integer i=1; i<subDomainKeyTree->numProcesses; i++) {
                    if ((sum + helper->integerBuffer[bodyIndex + offset]) >= (i*n) && sum < (i*n)) {
                        printf("[rank %i] new range: %lu\n", subDomainKeyTree->rank,
                               helper->keyTypeBuffer[bodyIndex + offset]);
                        subDomainKeyTree->range[i] = helper->keyTypeBuffer[bodyIndex + offset];
                    }
                }


                //printf("[rank %i] keyHistCounts[%i] = %i\n", s->rank, bodyIndex+offset, keyHistCounts[bodyIndex+offset]);
                atomicAdd(helper->integerVal, helper->integerBuffer[bodyIndex+offset]);
                offset += stride;
            }

        }

        __global__ void insertReceivedParticles(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                                DomainList *domainList, DomainList *lowestDomainList, int n, int m) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;

            //note: -1 used as "null pointer"
            //note: -2 used to lock a child (pointer)

            integer offset;
            bool newBody = true;

            real min_x, max_x;
#if DIM > 1
            real min_y, max_y;
#if DIM == 3
            real min_z, max_z;
#endif
#endif

            integer childPath;
            integer temp;

            bool isDomainList = false;

            offset = 0;

            bodyIndex += tree->toDeleteLeaf[0];

            while ((bodyIndex + offset) < tree->toDeleteLeaf[1] && (bodyIndex + offset) > tree->toDeleteLeaf[0]) {

                if ((bodyIndex + offset) % 100 == 0) {
                    printf("Inserting received particle %i: x = (%f, %f, %f) m = %f\n", bodyIndex + offset,
                           particles->x[bodyIndex + offset], particles->y[bodyIndex + offset],
                           particles->z[bodyIndex + offset], particles->mass[bodyIndex + offset]);
                }

                if (newBody) {

                    newBody = false;
                    isDomainList = false;

                    min_x = *tree->minX;
                    max_x = *tree->maxX;
#if DIM > 1
                    min_y = *tree->minY;
                    max_y = *tree->maxY;
#if DIM == 3
                    min_z = *tree->minZ;
                    max_z = *tree->maxZ;
#endif
#endif

                    temp = 0;
                    childPath = 0;

                    // find insertion point for body
                    if (particles->x[bodyIndex + offset] < 0.5 * (min_x + max_x)) { // x direction
                        childPath += 1;
                        max_x = 0.5 * (min_x + max_x);
                    }
                    else {
                        min_x = 0.5 * (min_x + max_x);
                    }
#if DIM > 1
                    if (particles->y[bodyIndex + offset] < 0.5 * (min_y + max_y)) { // y direction
                        childPath += 2;
                        max_y = 0.5 * (min_y + max_y);
                    }
                    else {
                        min_y = 0.5 * (min_y + max_y);
                    }
#if DIM == 3
                    if (particles->z[bodyIndex + offset] < 0.5 * (min_z + max_z)) {  // z direction
                        childPath += 4;
                        max_z = 0.5 * (min_z + max_z);
                    }
                    else {
                        min_z = 0.5 * (min_z + max_z);
                    }
#endif
#endif
                }

                int childIndex = tree->child[temp*POW_DIM + childPath];

                // traverse tree until hitting leaf node
                while (childIndex >= m /*&& childIndex < (8*m)*/) { //formerly n

                    isDomainList = false;

                    temp = childIndex;

                    childPath = 0;

                    // find insertion point for body
                    if (particles->x[bodyIndex + offset] < 0.5 * (min_x + max_x)) { // x direction
                        childPath += 1;
                        max_x = 0.5 * (min_x + max_x);
                    }
                    else {
                        min_x = 0.5 * (min_x + max_x);
                    }
#if DIM > 1
                    if (particles->y[bodyIndex + offset] < 0.5 * (min_y + max_y)) { // y direction
                        childPath += 2;
                        max_y = 0.5 * (min_y + max_y);
                    }
                    else {
                        min_y = 0.5 * (min_y + max_y);
                    }
#if DIM == 3
                    if (particles->z[bodyIndex + offset] < 0.5 * (min_z + max_z)) { // z direction
                        childPath += 4;
                        max_z = 0.5 * (min_z + max_z);
                    }
                    else {
                        min_z = 0.5 * (min_z + max_z);
                    }
#endif
#endif

                    for (int i=0; i<*domainList->domainListIndex; i++) {
                        if (temp == domainList->domainListIndices[i]) {
                            isDomainList = true;
                            break;
                        }
                    }

                    //TODO: !!!
                    if (/*true*/ !isDomainList) {
                        if (particles->mass[bodyIndex + offset] != 0) {
                            atomicAdd(&particles->x[temp], particles->mass[bodyIndex + offset] * particles->x[bodyIndex + offset]);
#if DIM > 1
                            atomicAdd(&particles->y[temp], particles->mass[bodyIndex + offset] * particles->y[bodyIndex + offset]);
#if DIM == 3
                            atomicAdd(&particles->z[temp], particles->mass[bodyIndex + offset] * particles->z[bodyIndex + offset]);
#endif
#endif
                        }
                        atomicAdd(&particles->mass[temp], particles->mass[bodyIndex + offset]);
                        //atomicAdd(&count[temp], 1); // do not count, since particles are just temporarily saved on this process
                    }
                    atomicAdd(&tree->count[temp], 1); // do not count, since particles are just temporarily saved on this process
                    childIndex = tree->child[POW_DIM*temp + childPath];
                }

                // if child is not locked
                if (childIndex != -2) {

                    int locked = temp * POW_DIM + childPath;

                    //lock
                    if (atomicCAS(&tree->child[locked], childIndex, -2) == childIndex) {

                        // check whether a body is already stored at the location
                        if (childIndex == -1) {
                            //insert body and release lock
                            tree->child[locked] = bodyIndex + offset;
                        }
                        else {
                            int patch = POW_DIM * m; //8*n
                            while (childIndex >= 0 && childIndex < n) {

                                //debug
                                //if (x[childIndex] == x[bodyIndex + offset]) {
                                //    printf("ATTENTION (shouldn't happen...): x[%i] = (%f, %f, %f) vs. x[%i] = (%f, %f, %f) | to_delete_leaf = (%i, %i)\n",
                                //           childIndex, x[childIndex], y[childIndex], z[childIndex], bodyIndex + offset,  x[bodyIndex + offset],
                                //           y[bodyIndex + offset], z[bodyIndex + offset], to_delete_leaf[0], to_delete_leaf[1]);
                                //}

                                //create a new cell (by atomically requesting the next unused array index)
                                int cell = atomicAdd(tree->index, 1);

                                patch = min(patch, cell);

                                if (patch != cell) {
                                    tree->child[POW_DIM * temp + childPath] = cell;
                                }

                                // TODO: remove!
                                // debug
                                if (particles->x[childIndex] == particles->x[bodyIndex + offset] &&
                                        particles->y[childIndex] == particles->y[bodyIndex + offset]) {
                                    printf("[rank %i] ATTENTION!!! %i vs. %i ((%f, %f, %f) vs (%f, %f, %f))\n", subDomainKeyTree->rank,
                                           childIndex, bodyIndex + offset,
                                           particles->x[childIndex], particles->y[childIndex], particles->z[childIndex],
                                           particles->x[bodyIndex+offset], particles->y[bodyIndex+offset], particles->z[bodyIndex+offset]);
                                    break;
                                }
                                // end: debug

                                // insert old/original particle
                                childPath = 0;
                                if (particles->x[childIndex] < 0.5 * (min_x + max_x)) { childPath += 1; }
#if DIM > 1
                                if (particles->y[childIndex] < 0.5 * (min_y + max_y)) { childPath += 2; }
#if DIM == 3
                                if (particles->z[childIndex] < 0.5 * (min_z + max_z)) { childPath += 4; }
#endif
#endif

                                particles->x[cell] += particles->mass[childIndex] * particles->x[childIndex];
#if DIM > 1
                                particles->y[cell] += particles->mass[childIndex] * particles->y[childIndex];
#if DIM == 3
                                particles->z[cell] += particles->mass[childIndex] * particles->z[childIndex];
#endif
#endif

                                particles->mass[cell] += particles->mass[childIndex];
                                // do not count, since particles are just temporarily saved on this process
                                tree->count[cell] += tree->count[childIndex];

                                tree->child[POW_DIM * cell + childPath] = childIndex;

                                tree->start[cell] = -1; //TODO: resetting start needed in insertReceivedParticles()?

                                // insert new particle
                                temp = cell;
                                childPath = 0;

                                // find insertion point for body
                                if (particles->x[bodyIndex + offset] < 0.5 * (min_x + max_x)) {
                                    childPath += 1;
                                    max_x = 0.5 * (min_x + max_x);
                                } else {
                                    min_x = 0.5 * (min_x + max_x);
                                }
#if DIM > 1
                                if (particles->y[bodyIndex + offset] < 0.5 * (min_y + max_y)) {
                                    childPath += 2;
                                    max_y = 0.5 * (min_y + max_y);
                                } else {
                                    min_y = 0.5 * (min_y + max_y);
                                }
#if DIM == 3
                                if (particles->z[bodyIndex + offset] < 0.5 * (min_z + max_z)) {
                                    childPath += 4;
                                    max_z = 0.5 * (min_z + max_z);
                                } else {
                                    min_z = 0.5 * (min_z + max_z);
                                }
#endif
#endif

                                // COM / preparing for calculation of COM
                                if (particles->mass[bodyIndex + offset] != 0) {
                                    particles->x[cell] += particles->mass[bodyIndex + offset] * particles->x[bodyIndex + offset];
#if DIM > 1
                                    particles->y[cell] += particles->mass[bodyIndex + offset] * particles->y[bodyIndex + offset];
#if DIM == 3
                                    particles->z[cell] += particles->mass[bodyIndex + offset] * particles->z[bodyIndex + offset];
#endif
#endif
                                    particles->mass[cell] += particles->mass[bodyIndex + offset];
                                }
                                // do not count, since particles are just temporarily saved on this process
                                tree->count[cell] += tree->count[bodyIndex + offset];

                                childIndex = tree->child[POW_DIM * temp + childPath];
                            }

                            tree->child[POW_DIM * temp + childPath] = bodyIndex + offset;

                            __threadfence();  // written to global memory arrays (child, x, y, mass) thus need to fence
                            tree->child[locked] = patch;
                        }
                        offset += stride;
                        newBody = true;
                    }
                    else {

                    }
                }
                else {

                }
                __syncthreads();
            }

        }

        __global__ void centreOfMassReceivedParticles(Particles *particles, integer *startIndex,
                                                            integer *endIndex, int n) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;

            //note: most of it already done within buildTreeKernel
            bodyIndex += *startIndex;

            while ((bodyIndex + offset) < *endIndex) {

                //if (particles->mass[bodyIndex + offset] == 0) {
                //    printf("centreOfMassKernel: mass = 0 (%i)!\n", bodyIndex + offset);
                //}

                if (particles->mass[bodyIndex + offset] != 0) {
                    particles->x[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
                    particles->y[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
                    particles->z[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
                }

                offset += stride;
            }

        }

        __global__ void repairTree(Tree *tree, Particles *particles, DomainList *domainList, int n, int m) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            if (bodyIndex + offset == 0) {
                *tree->index = tree->toDeleteNode[0];
            }

            offset = tree->toDeleteLeaf[0];
            //delete inserted leaves
            while ((bodyIndex + offset) >= tree->toDeleteLeaf[0] && (bodyIndex + offset) < tree->toDeleteLeaf[1]) {
                for (int i=0; i<POW_DIM; i++) {
                    tree->child[(bodyIndex + offset)*POW_DIM + i] = -1;
                }
                tree->count[bodyIndex + offset] = 1;

                particles->x[bodyIndex + offset] = 0;
                particles->vx[bodyIndex + offset] = 0;
                particles->ax[bodyIndex + offset] = 0;
#if DIM > 1
                particles->y[bodyIndex + offset] = 0;
                particles->vy[bodyIndex + offset] = 0;
                particles->ay[bodyIndex + offset] = 0;
#if DIM == 3
                particles->z[bodyIndex + offset] = 0;
                particles->vz[bodyIndex + offset] = 0;
                particles->az[bodyIndex + offset] = 0;
#endif
#endif
                particles->mass[bodyIndex + offset] = 0;
                tree->start[bodyIndex + offset] = -1;
                //sorted[bodyIndex + offset] = 0;

                offset += stride;
            }

            offset = tree->toDeleteNode[0]; //0;
            //delete inserted cells
            while ((bodyIndex + offset) >= tree->toDeleteNode[0] && (bodyIndex + offset) < tree->toDeleteNode[1]) {
                for (int i=0; i<POW_DIM; i++) {
                    tree->child[(bodyIndex + offset)*POW_DIM + i] = -1;
                }
                tree->count[bodyIndex + offset] = 0;
                particles->x[bodyIndex + offset] = 0;
                particles->vx[bodyIndex + offset] = 0;
                particles->ax[bodyIndex + offset] = 0;
#if DIM > 1
                particles->y[bodyIndex + offset] = 0;
                particles->vy[bodyIndex + offset] = 0;
                particles->ay[bodyIndex + offset] = 0;
#if DIM == 3
                particles->z[bodyIndex + offset] = 0;
                particles->vz[bodyIndex + offset] = 0;
                particles->az[bodyIndex + offset] = 0;
#endif
#endif
                particles->mass[bodyIndex + offset] = 0;
                tree->start[bodyIndex + offset] = -1;
                //sorted[bodyIndex + offset] = 0;

                offset += stride;
            }
        }

        real Launch::zeroDomainListNodes(Particles *particles, DomainList *domainList, DomainList *lowestDomainList) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::zeroDomainListNodes, particles, domainList,
                                lowestDomainList);
        }

        real Launch::prepareLowestDomainExchange(Particles *particles, DomainList *lowestDomainList,
                                                 Helper *helper, Entry::Name entry) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::prepareLowestDomainExchange, particles,
                                lowestDomainList, helper, entry);
        }

        real Launch::updateLowestDomainListNodes(Particles *particles, DomainList *lowestDomainList,
                                                 Helper *helper, Entry::Name entry) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::updateLowestDomainListNodes, particles,
                                lowestDomainList, helper, entry);

        }

        real Launch::compLowestDomainListNodes(Particles *particles, DomainList *lowestDomainList) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::compLowestDomainListNodes, particles,
                                lowestDomainList);
        }

        real Launch::compLocalPseudoParticles(Tree *tree, Particles *particles, DomainList *domainList, int n) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::compLocalPseudoParticles, tree, particles,
                                domainList, n);
        }

        real Launch::compDomainListPseudoParticles(Tree *tree, Particles *particles, DomainList *domainList,
                                                   DomainList *lowestDomainList, int n) {
            ExecutionPolicy executionPolicy(256, 1);
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::compDomainListPseudoParticles, tree,
                                particles, domainList, lowestDomainList, n);
        }

        real Launch::computeForces(Tree *tree, Particles *particles, integer n, integer m, integer blockSize,
                                   integer warp, integer stackSize) {

            //TODO: check shared memory size
            //size_t sharedMemory = (sizeof(real)+sizeof(integer))*stackSize*blockSize/warp;
            size_t sharedMemory = 2*sizeof(real)*stackSize*blockSize/warp;
            ExecutionPolicy executionPolicy(256, 256, sharedMemory);
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::computeForces, tree, particles, n, m,
                                blockSize, warp, stackSize);
        }

        real Launch::update(Particles *particles, integer n, real dt, real d) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::update, particles, n, dt, d);
        }

        real Launch::symbolicForce(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                           DomainList *domainList, Helper *helper,
                           real diam, real theta_, integer n, integer m, integer relevantIndex,
                           Curve::Type curveType) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::symbolicForce, subDomainKeyTree, tree,
                                particles, domainList, helper, diam, theta_, n, m, relevantIndex, curveType);
        }

        real Launch::compTheta(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                       DomainList *domainList, Helper *helper, Curve::Type curveType) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::compTheta, subDomainKeyTree, tree, particles,
                                domainList, helper, curveType);
        }

        real Launch::createKeyHistRanges(Helper *helper, integer bins) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::createKeyHistRanges, helper, bins);
        }

        real Launch::keyHistCounter(Tree *tree, Particles *particles, SubDomainKeyTree *subDomainKeyTree,
                            Helper *helper, int bins, int n, Curve::Type curveType) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::keyHistCounter, tree, particles,
                                subDomainKeyTree, helper, bins, n, curveType);
        }

        real Launch::calculateNewRange(SubDomainKeyTree *subDomainKeyTree, Helper *helper, int bins, int n,
                               Curve::Type curveType) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::calculateNewRange, subDomainKeyTree, helper,
                                bins, n, curveType);
        }

        real Launch::insertReceivedParticles(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                     DomainList *domainList, DomainList *lowestDomainList, int n, int m) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::insertReceivedParticles, subDomainKeyTree,
                                tree, particles, domainList, lowestDomainList, n, m);
        }

        real Launch::centreOfMassReceivedParticles(Particles *particles, integer *startIndex,
                                           integer *endIndex, int n) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::centreOfMassReceivedParticles,
                                particles, startIndex, endIndex, n);
        }

        real Launch::repairTree(Tree *tree, Particles *particles, DomainList *domainList, int n, int m) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::repairTree, tree, particles, domainList,
                                n, m);
        }

    }
}
