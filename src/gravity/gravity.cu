#include "hip/hip_runtime.h"
#include "../../include/gravity/gravity.cuh"
#include "../../include/cuda_utils/cuda_launcher.cuh"

namespace Gravity {

    namespace Kernel {

        __global__ void globalCOM(Tree *tree, Particles *particles, real com[DIM]) {

            real mass = 0;
            for (int i=0; i<DIM; i++) {
                com[i] = 0;
            }
            for (int i=0; i<POW_DIM; i++) {
                if (tree->child[i] != -1) {
                    mass += particles->mass[tree->child[i]];
                    com[0] += particles->weightedEntry(tree->child[i], Entry::x);
#if DIM > 1
                    com[1] += particles->weightedEntry(tree->child[i], Entry::y);
#if DIM == 3
                    com[2] += particles->weightedEntry(tree->child[i], Entry::z);
#endif
#endif
                }
            }

            if (mass > 0) {
                com[0] /= mass;
#if DIM > 1
                com[1] /= mass;
#if DIM == 3
                com[2] /= mass;
#endif
#endif
            }

        }

        __global__ void collectSendIndices(Tree *tree, Particles *particles, integer *sendIndices,
                                           integer *particles2Send, integer *pseudoParticles2Send,
                                           integer *pseudoParticlesLevel,
                                           integer *particlesCount, integer *pseudoParticlesCount,
                                           integer n, integer length, Curve::Type curveType) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;

            integer particleInsertIndex;
            integer pseudoParticleInsertIndex;

            while ((bodyIndex + offset) < length) {

                if (sendIndices[bodyIndex + offset] == 1) {

                    // it is a particle
                    if (bodyIndex + offset < n) {
                        particleInsertIndex = atomicAdd(particlesCount, 1);
                        particles2Send[particleInsertIndex] = bodyIndex + offset;
                    }
                    // it is a pseudo-particle
                    else {
                        pseudoParticleInsertIndex = atomicAdd(pseudoParticlesCount, 1);
                        pseudoParticles2Send[pseudoParticleInsertIndex] = bodyIndex + offset;
                        pseudoParticlesLevel[pseudoParticleInsertIndex] = particles->level[bodyIndex + offset];
                        //printf("pseudo-particle level to be sent: %i (%i)\n", particles->level[bodyIndex + offset],
                        //       bodyIndex + offset);
                        //pseudoParticlesLevel[pseudoParticleInsertIndex] = tree->getTreeLevel(particles,
                        //                                                                     bodyIndex + offset,
                        //                                                                     MAX_LEVEL, curveType);

                        // debug
                        //if (pseudoParticlesLevel[pseudoParticleInsertIndex] == -1) {
                        //    printf("level = -1 within collectSendIndices for index: %i\n", bodyIndex + offset);
                        //}
                        // end: debug
                    }
                }
                __threadfence();
                offset += stride;
            }
        }

        __global__ void testSendIndices(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                        integer *sendIndices, integer *markedSendIndices,
                                        integer *levels, Curve::Type curveType, integer length) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;

            integer key;
            integer temp;
            integer childIndex;
            integer childPath;

            bool available = false;

            real min_x, max_x;
#if DIM > 1
            real min_y, max_y;
#if DIM == 3
            real min_z, max_z;
#endif
#endif

            /*while ((bodyIndex + offset) < length) {

                printf("x[%i] = (%f, %f, %f) %f\n", sendIndices[bodyIndex + offset], particles->x[sendIndices[bodyIndex + offset]],
                       particles->y[sendIndices[bodyIndex + offset]], particles->z[sendIndices[bodyIndex + offset]],
                       particles->mass[sendIndices[bodyIndex + offset]]);

                offset += stride;
            }*/

            //while ((bodyIndex + offset) < length) {
            //    if (particles->x[sendIndices[bodyIndex + offset]] == 0.f &&
            //        particles->y[sendIndices[bodyIndex + offset]] == 0.f &&
            //        particles->z[sendIndices[bodyIndex + offset]] == 0.f &&
            //        particles->mass[sendIndices[bodyIndex + offset]]) {
            //
            //    }
            //    offset += stride;
            //}

            // ///////////////////////////////////////////////////////////////////////////////////

            //if (bodyIndex == 0) {
            //    for (int i = 0; i<10; i++) {
            //        printf("sendIndices[%i] = %i (length = %i)\n", length - 1 + i, sendIndices[length -1 + i], length);
            //    }
            //}

            //if (bodyIndex == 0) {
            //    integer i=0;
            //    for (int i = 0; i<30000; i++) {
            //        if (markedSendIndices[100000 + i] == 1) {
            //            printf("[rank %i] markedSendIndices[%i] = %i!\n", subDomainKeyTree->rank, 100000 + i, markedSendIndices[100000 + i]);
            //            break;
            //        }
            //    }
            //}

            while ((bodyIndex + offset) < length) {

                //printf("index = %i sendIndex = %i level = %i\n", bodyIndex + offset, sendIndices[bodyIndex + offset],
                //       levels[bodyIndex + offset]);

                min_x = *tree->minX;
                max_x = *tree->maxX;
#if DIM > 1
                min_y = *tree->minY;
                max_y = *tree->maxY;
#if DIM == 3
                min_z = *tree->minZ;
                max_z = *tree->maxZ;
#endif
#endif

                available = false;

                childIndex = 0;
                if (levels[bodyIndex + offset] > 3) {
                    //key = tree->getParticleKey(particles, bodyIndex + offset + tree->toDeleteNode[0], MAX_LEVEL,
                    //                           curveType);

                    //printf("level = %i\n", levels[bodyIndex + offset]);

                    childIndex = 0;

                    for (int j = 0; j < levels[bodyIndex + offset] - 1; j++) {

                        temp = childIndex;

                        childPath = 0;
                        if (particles->x[sendIndices[bodyIndex + offset]] < 0.5 * (min_x + max_x)) {
                            childPath += 1;
                            max_x = 0.5 * (min_x + max_x);
                        } else {
                            min_x = 0.5 * (min_x + max_x);
                        }
#if DIM > 1
                        if (particles->y[sendIndices[bodyIndex + offset]] < 0.5 * (min_y + max_y)) {
                            childPath += 2;
                            max_y = 0.5 * (min_y + max_y);
                        } else {
                            min_y = 0.5 * (min_y + max_y);
                        }
#if DIM == 3
                        if (particles->z[sendIndices[bodyIndex + offset]] < 0.5 * (min_z + max_z)) {
                            childPath += 4;
                            max_z = 0.5 * (min_z + max_z);
                        } else {
                            min_z = 0.5 * (min_z + max_z);
                        }
#endif
#endif
                        //printf("childIndex = %i\n", childIndex);
                        childIndex = tree->child[POW_DIM * temp + childPath];
                        if (bodyIndex + offset == 0) {
                            printf("tree->child[POW_DIM * %i + %i] = %i (%i)\n", temp, childPath, tree->child[POW_DIM * temp + childPath], sendIndices[bodyIndex + offset]);
                        }
                    }


                    for (int i = 0; i < length; i++) {
                        if (temp == sendIndices[i]) {
                            available = true;
                            break;
                        }
                    }

                    if (!available) {
                        //integer a = -1;
                        //markedSendIndices[childIndex] = a;
                        printf("[rank %i] %i (relevant son: %i) NOT Available sendIndices[%i] = %i, [%i] = %i)!\n",
                               subDomainKeyTree->rank, temp, childIndex,
                               childIndex, markedSendIndices[childIndex], temp, markedSendIndices[temp]);
                        assert(0);
                    }

                    //if (childIndex != sendIndices[bodyIndex + offset]) {
                        //printf("ATTENTION childIndex != bodyIndex level = %i (%i != %i) (%f, %f, %f)!\n", levels[bodyIndex + offset], childIndex, sendIndices[bodyIndex + offset],
                        //       particles->x[sendIndices[bodyIndex + offset]], particles->y[sendIndices[bodyIndex + offset]],
                        //       particles->z[sendIndices[bodyIndex + offset]]);
                    //} else {
                        //printf("--\n");
                    //}
                }

                offset += stride;
            }
        }

        __global__ void zeroDomainListNodes(Particles *particles, DomainList *domainList,
                                            DomainList *lowestDomainList) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;
            integer domainIndex;
            bool zero;

            while ((bodyIndex + offset) < *domainList->domainListIndex) {
                zero = true;
                domainIndex = domainList->domainListIndices[bodyIndex + offset];
                //for (int i=0; i<*lowestDomainList->domainListIndex; i++) {
                //    if (domainIndex == lowestDomainList->domainListIndices[i]) {
                //        zero = false;
                //        break;
                //    }
                //}

                if (particles->nodeType[domainIndex] == 2) {
                    zero = false;
                }

                if (zero) {
                    //printf("zero %i\n", domainIndex);
                    particles->x[domainIndex] = (real)0;
#if DIM > 1
                    particles->y[domainIndex] = (real)0;
#if DIM == 3
                    particles->z[domainIndex] = (real)0;
#endif
#endif

                    particles->mass[domainIndex] = (real)0;
                }
                /*
                else {
                    //printf("domainIndex = %i *= mass = %f\n", domainIndex, particles->mass[domainIndex]);
                    particles->x[domainIndex] *= particles->mass[domainIndex];
#if DIM > 1
                    particles->y[domainIndex] *= particles->mass[domainIndex];
#if DIM == 3
                    particles->z[domainIndex] *= particles->mass[domainIndex];
#endif
#endif
                }
                */


                offset += stride;
            }

        }

        __global__ void prepareLowestDomainExchange(Particles *particles, DomainList *lowestDomainList,
                                                    Helper *helper, Entry::Name entry) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;
            integer index;
            integer lowestDomainIndex;

            //copy x, y, z, mass of lowest domain list nodes into arrays
            //sorting using cub (not here)
            while ((bodyIndex + offset) < *lowestDomainList->domainListIndex) {
                lowestDomainIndex = lowestDomainList->domainListIndices[bodyIndex + offset];
                if (lowestDomainIndex >= 0) {
                    switch (entry) {
                        case Entry::x: {
                            helper->realBuffer[bodyIndex + offset] = particles->x[lowestDomainIndex];
                        } break;
#if DIM > 1
                        case Entry::y: {
                            helper->realBuffer[bodyIndex + offset] = particles->y[lowestDomainIndex];
                        } break;
#if DIM == 3
                        case Entry::z: {
                            helper->realBuffer[bodyIndex + offset] = particles->z[lowestDomainIndex];
                        } break;
#endif
#endif
                        case Entry::mass: {
                            helper->realBuffer[bodyIndex + offset] = particles->mass[lowestDomainIndex];
                        } break;
                        default:
                            printf("prepareLowestDomainExchange(): Not available!\n");
                    }
                }
                offset += stride;
            }
        }

        __global__ void updateLowestDomainListNodes(Particles *particles, DomainList *lowestDomainList,
                                                    Helper *helper, Entry::Name entry) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            integer originalIndex = -1;

            while ((bodyIndex + offset) < *lowestDomainList->domainListIndex) {
                originalIndex = -1;
                for (int i = 0; i < *lowestDomainList->domainListIndex; i++) {
                    if (lowestDomainList->sortedDomainListKeys[bodyIndex + offset] ==
                        lowestDomainList->domainListKeys[i]) {
                        originalIndex = i;
                    }
                }

                if (originalIndex == -1) {
                    cudaTerminate("ATTENTION: originalIndex = -1 (index = %i)!\n",
                           lowestDomainList->sortedDomainListKeys[bodyIndex + offset]);
                }

                switch (entry) {
                    case Entry::x: {
                        particles->x[lowestDomainList->domainListIndices[originalIndex]] =
                                helper->realBuffer[DOMAIN_LIST_SIZE + bodyIndex + offset];
                    } break;
#if DIM > 1
                    case Entry::y: {
                        particles->y[lowestDomainList->domainListIndices[originalIndex]] =
                                helper->realBuffer[DOMAIN_LIST_SIZE + bodyIndex + offset];
                    } break;
#if DIM == 3
                    case Entry::z: {
                        particles->z[lowestDomainList->domainListIndices[originalIndex]] =
                                helper->realBuffer[DOMAIN_LIST_SIZE + bodyIndex + offset];
                    } break;
#endif
#endif
                    case Entry::mass: {
                        particles->mass[lowestDomainList->domainListIndices[originalIndex]] =
                                helper->realBuffer[DOMAIN_LIST_SIZE + bodyIndex + offset];
                    } break;
                    default: {
                        printf("Entry not available!\n");
                    }
                }

                offset += stride;
            }
        }

        __global__ void compLowestDomainListNodes(Tree *tree, Particles *particles, DomainList *lowestDomainList) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;
            integer lowestDomainIndex;
            bool divide;

            while ((bodyIndex + offset) < *lowestDomainList->domainListIndex) {

                //divide = false;
                lowestDomainIndex = lowestDomainList->domainListIndices[bodyIndex + offset];

                //for (int child=0; child<POW_DIM; child++) {
                //    if (tree->child[POW_DIM * lowestDomainIndex + child] != -1) {
                //        printf("lowestDomainIndex: tree->child[8 * %i + %i] = %i\n", lowestDomainIndex, child,
                //               tree->child[POW_DIM * lowestDomainIndex + child]);
                //        divide = true;
                //        break;
                //    }
                //}

                //if (particles->mass[lowestDomainIndex] != (real)0) {
                //if (particles->mass[lowestDomainIndex] > (real)0) {
                if (/*divide && */particles->mass[lowestDomainIndex] > (real)0) {

#if DIM == 3
                    //printf("lowestDomainIndex: %i (%f, %f, %f) %f\n", lowestDomainIndex, particles->x[lowestDomainIndex],
                    //       particles->y[lowestDomainIndex], particles->z[lowestDomainIndex], particles->mass[lowestDomainIndex]);
#endif

                    /*particles->x[lowestDomainIndex] /= particles->mass[lowestDomainIndex];
#if DIM > 1
                    particles->y[lowestDomainIndex] /= particles->mass[lowestDomainIndex];
#if DIM == 3
                    particles->z[lowestDomainIndex] /= particles->mass[lowestDomainIndex];
#endif
#endif
                     */

                    //if (std::isnan(particles->x[lowestDomainIndex])) {
                    //    printf("NAN! within compLowestDomainListNodes for %i: x = %f, m = %f\n", lowestDomainIndex, particles->x[lowestDomainIndex],
                    //           particles->mass[lowestDomainIndex]);
                    //    assert(0);
                    //}

#if DIM == 3
                    //printf("lowestDomainIndex: %i (%f, %f, %f) %f\n", lowestDomainIndex, particles->x[lowestDomainIndex],
                    //       particles->y[lowestDomainIndex], particles->z[lowestDomainIndex], particles->mass[lowestDomainIndex]);
#endif
                }

                //printf("lowestDomainIndex = %i (%f, %f, %f) %f\n", lowestDomainIndex, particles->x[lowestDomainIndex],
                //       particles->y[lowestDomainIndex], particles->z[lowestDomainIndex], particles->mass[lowestDomainIndex]);

                offset += stride;
            }
        }

        __global__ void compLocalPseudoParticles(Tree *tree, Particles *particles, DomainList *domainList, int n) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;
            bool isDomainList;

            bodyIndex += n;

            while (bodyIndex + offset < *tree->index) {
                isDomainList = false;

                //for (integer i=0; i<*domainList->domainListIndex; i++) {
                //    if ((bodyIndex + offset) == domainList->domainListIndices[i]) {
                //        isDomainList = true; // hence do not insert
                //        break;
                //    }
                //}
                if (particles->nodeType[bodyIndex + offset] >= 1) {
                    isDomainList = true;
                }

                if (/*particles->mass[bodyIndex + offset] != 0 && */!isDomainList) {
                    particles->x[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
#if DIM > 1
                    particles->y[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
#if DIM == 3
                    particles->z[bodyIndex + offset] /= particles->mass[bodyIndex + offset];
#endif
#endif
                }

                offset += stride;
            }
        }

        __global__ void compDomainListPseudoParticlesPerLevel(Tree *tree, Particles *particles, DomainList *domainList,
                                                              DomainList *lowestDomainList, int n, int level) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset;

            integer domainIndex;
            //integer level = MAX_LEVEL; // max level
            bool compute;

            real totalMass, x_mass, y_mass, z_mass;
            int childToLookAt = 5;
            int lowestDomainIndex, childLowestDomain;

            offset = 0;
            compute = true;
            while ((bodyIndex + offset) < *domainList->domainListIndex) {
                compute = true;
                domainIndex = domainList->domainListIndices[bodyIndex + offset];
                if (particles->nodeType[domainIndex] == 2) {
                    compute = false;
                }
                //for (int i=0; i<*lowestDomainList->domainListIndex; i++) {
                //    if (domainIndex == lowestDomainList->domainListIndices[i]) {
                //        compute = false;
                //        /*lowestDomainIndex = domainIndex;
                //        while (lowestDomainIndex != -1) {
                //            //childLowestDomain;
                //            printf("(%lu) lowestDomainIndex: %i (%e, %e, %e) %e\n", lowestDomainList->domainListKeys[i],
                //                   lowestDomainIndex,
                //                   particles->x[lowestDomainIndex], particles->y[lowestDomainIndex],
                //                   particles->z[lowestDomainIndex], particles->mass[lowestDomainIndex]);
                //            totalMass = 0.;
                //            x_mass = 0.;
                //            y_mass = 0.;
                //            z_mass = 0.;
                //            for (int i_child = 0; i_child < POW_DIM; i_child++) {
                //                childLowestDomain = tree->child[POW_DIM * lowestDomainIndex + i_child];
                //                if (childLowestDomain != -1) {
                //                    //printf("totalMass += %e\n", particles->mass[childLowestDomain]);
                //                    totalMass += particles->mass[childLowestDomain];
                //                    x_mass += particles->mass[childLowestDomain] * particles->x[childLowestDomain];
                //                    y_mass += particles->mass[childLowestDomain] * particles->y[childLowestDomain];
                //                    z_mass += particles->mass[childLowestDomain] * particles->z[childLowestDomain];
                //                    //if (totalMass > 0.) {
                //                    //    printf("totalMass = %e > %e\n", totalMass, particles->mass[lowestDomainIndex]);
                //                    //}
                //                }
                //                //if (particles->mass[childLowestDomain] > 0.) {
                //                //    printf("!= 0: %e\n", particles->mass[childLowestDomain]);
                //                //}
                //                printf("(%lu) lowestDomainIndex: %i child #%i -> %i (%e, %e, %e) %e\n",
                //                       lowestDomainList->domainListKeys[i],
                //                       lowestDomainIndex, i_child, childLowestDomain,
                //                       particles->x[childLowestDomain], particles->y[childLowestDomain],
                //                       particles->z[childLowestDomain], particles->mass[childLowestDomain]);
                //            }
                //            x_mass /= totalMass;
                //            y_mass /= totalMass;
                //            z_mass /= totalMass;
                //            if (totalMass > (particles->mass[lowestDomainIndex] + 1e-7)) { //||
                //                //x_mass < (particles->x[lowestDomainIndex] - 1e-7) ||
                //                //y_mass < (particles->y[lowestDomainIndex] - 1e-7) ||
                //                //z_mass < (particles->z[lowestDomainIndex] - 1e-7)) {
                //                //if (totalMass > 0.) {
                //                printf("totalMass = %e > %e\n", totalMass, particles->mass[lowestDomainIndex]);
                //                //}
                //                assert(0);
                //            }
                //            lowestDomainIndex = tree->child[POW_DIM * lowestDomainIndex + childToLookAt];
                //        }*/
                //        break;
                //    }
                //}
                if (compute && domainList->domainListLevels[bodyIndex + offset] == level) {
                    // do the calculation
                    //particles->x[domainIndex] = 0.;
                    //particles->y[domainIndex] = 0.;
                    //particles->z[domainIndex] = 0.;
                    //particles->mass[domainIndex] = 0.;
                    for (int i=0; i<POW_DIM; i++) {
                        particles->x[domainIndex] += particles->x[tree->child[POW_DIM*domainIndex + i]] *
                                                     particles->mass[tree->child[POW_DIM*domainIndex + i]];
#if DIM > 1
                        particles->y[domainIndex] += particles->y[tree->child[POW_DIM*domainIndex + i]] *
                                                     particles->mass[tree->child[POW_DIM*domainIndex + i]];
#if DIM == 3
                        particles->z[domainIndex] += particles->z[tree->child[POW_DIM*domainIndex + i]] *
                                                     particles->mass[tree->child[POW_DIM*domainIndex + i]];
#endif
#endif
                        particles->mass[domainIndex] += particles->mass[tree->child[POW_DIM*domainIndex + i]];
                    }

                    if (particles->mass[domainIndex] != 0.f) {
                        particles->x[domainIndex] /= particles->mass[domainIndex];
#if DIM > 1
                        particles->y[domainIndex] /= particles->mass[domainIndex];
#if DIM == 3
                        particles->z[domainIndex] /= particles->mass[domainIndex];
#endif
#endif
                    }
                }
                offset += stride;
            }
            __syncthreads();
        }

        __global__ void compDomainListPseudoParticles(Tree *tree, Particles *particles, DomainList *domainList,
                                                      DomainList *lowestDomainList, int n) {
            //calculate position (center of mass) and mass for domain list nodes
            //Problem: start with "deepest" nodes
            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset;

            integer domainIndex;
            integer level = MAX_LEVEL; // max level
            bool compute;

            // go from max level to level=0
            while (level >= 0) {
                offset = 0;
                compute = true;
                while ((bodyIndex + offset) < *domainList->domainListIndex) {
                    compute = true;
                    domainIndex = domainList->domainListIndices[bodyIndex + offset];
                    if (particles->nodeType[domainIndex] == 2) {
                        compute = false;
                    }
                    //for (int i=0; i<*lowestDomainList->domainListIndex; i++) {
                    //    if (domainIndex == lowestDomainList->domainListIndices[i]) {
                    //        compute = false;
                    //    }
                    //}
                    if (compute && domainList->domainListLevels[bodyIndex + offset] == level) {
                        // do the calculation
                        for (int i=0; i<POW_DIM; i++) {
                            particles->x[domainIndex] += particles->x[tree->child[POW_DIM*domainIndex + i]] *
                                    particles->mass[tree->child[POW_DIM*domainIndex + i]];
#if DIM > 1
                            particles->y[domainIndex] += particles->y[tree->child[POW_DIM*domainIndex + i]] *
                                    particles->mass[tree->child[POW_DIM*domainIndex + i]];
#if DIM == 3
                            particles->z[domainIndex] += particles->z[tree->child[POW_DIM*domainIndex + i]] *
                                    particles->mass[tree->child[POW_DIM*domainIndex + i]];
#endif
#endif
                            particles->mass[domainIndex] += particles->mass[tree->child[POW_DIM*domainIndex + i]];
                        }

                        if (particles->mass[domainIndex] != 0.f) {
                            particles->x[domainIndex] /= particles->mass[domainIndex];
#if DIM > 1
                            particles->y[domainIndex] /= particles->mass[domainIndex];
#if DIM == 3
                            particles->z[domainIndex] /= particles->mass[domainIndex];
#endif
#endif
                        }
                    }
                    offset += stride;
                }
                __syncthreads();
                level--;
            }
        }

        __global__ void computeForces_v1(Tree *tree, Particles *particles, real radius, integer n, integer m,
                                         SubDomainKeyTree *subDomainKeyTree, real theta, real smoothing) {

            register int i, ii;
            int child, nodeIndex, childNumber, depth;

            real px, ax, dx, f, distance;
#if DIM > 1
            real py, ay, dy;
#if DIM == 3
            real pz, az, dz;
#endif
#endif
            real sml;
            real thetasq = theta*theta;

            int currentNodeIndex[MAX_DEPTH];
            int currentChildNumber[MAX_DEPTH];

            __shared__ volatile real cellSize[MAX_DEPTH];

            if (threadIdx.x == 0) {
                cellSize[0] = 4.0 * radius * radius; //4.0 * radius * radius; //TODO: original one is 4.0 * radi...
#pragma unroll
                for (i = 1; i < MAX_DEPTH; i++) {
                    cellSize[i] = cellSize[i - 1] * 0.25;
                }
            }

            __syncthreads();

            for (ii = threadIdx.x + blockIdx.x * blockDim.x; ii < n; ii += blockDim.x * gridDim.x) {

                i = tree->sorted[ii]; //i = ii;

                px = particles->x[i];
#if DIM > 1
                py = particles->y[i];
#if DIM == 3
                pz = particles->z[i];
#endif
#endif
                //particles->ax[i] = 0.0;
                particles->g_ax[i] = 0.0;
#if DIM > 1
                //particles->ay[i] = 0.0;
                particles->g_ay[i] = 0.0;
#if DIM == 3
                //particles->az[i] = 0.0;
                particles->g_az[i] = 0.0;
#endif
#endif
                ax = 0.0;
#if DIM > 1
                ay = 0.0;
#if DIM == 3
                az = 0.0;
#endif
#endif

                // start at root
                depth = 1;
                currentNodeIndex[depth] = 0;
                currentChildNumber[depth] = 0;

                do {
                    childNumber = currentChildNumber[depth];
                    nodeIndex = currentNodeIndex[depth];

                    while(childNumber < POW_DIM) {
                        do {
                            child = tree->child[POW_DIM * nodeIndex + childNumber];
                            childNumber++;
                        } while(child == -1 && childNumber < POW_DIM);

                        if (child != -1 && child != i) { // dont do self-gravity with yourself!
                            dx = particles->x[child] - px;
                            distance = dx*dx + smoothing; //150329404.287723; //(0.0317 * 0.0317); //0.025;
#if DIM > 1
                            dy = particles->y[child] - py;
                            distance += dy*dy;
#endif
#if DIM == 3
                            dz = particles->z[child] - pz;
                            distance += dz*dz;
#endif
                            // if child is leaf or far away
                            if (child < n || distance * thetasq > cellSize[depth]) {
                                distance = cuda::math::sqrt(distance);
#if SI_UNITS
                                f = Constants::G * particles->mass[child] / (distance * distance * distance);
#else
                                f = particles->mass[child] / (distance * distance * distance);
#endif

                                ax += f*dx;
#if DIM > 1
                                ay += f*dy;
#if DIM == 3
                                az += f*dz;
#endif
#endif
                                //TODO: some flag for calculating potential energy
                                // gravitational potential energy
                                //particles->u[i] -= 0.5 * (particles->mass[child] * particles->mass[i])/distance;
                                // end: gravitational potential energy
                            } else {
                                // put child on stack
                                currentChildNumber[depth] = childNumber;
                                currentNodeIndex[depth] = nodeIndex;
                                depth++;
                                if (depth == MAX_DEPTH) {
                                    cudaTerminate("depth = %i >= MAX_DEPTH = %i\n", depth, MAX_DEPTH);
                                }
                                childNumber = 0;
                                nodeIndex = child;
                            }
                        }
                    }
                    depth--;
                } while(depth > 0);

                //particles->ax[i] = ax;
                particles->g_ax[i] = ax;
#if DIM > 1
                //particles->ay[i] = ay;
                particles->g_ay[i] = ay;
#if DIM == 3
                //particles->az[i] = az;
                particles->g_az[i] = az;
#endif
#endif
            }
        }

        __global__ void computeForces_v1_1(Tree *tree, Particles *particles, real radius, integer n, integer m,
                                         SubDomainKeyTree *subDomainKeyTree, real theta, real smoothing) {

            integer i, ii, child, nodeIndex, childNumber, depth;

            real px, ax, dx, f, distance;
#if DIM > 1
            real py, ay, dy;
#if DIM == 3
            real pz, az, dz;
#endif
#endif

            real sml;
            real thetasq = theta*theta;

            integer currentNodeIndex[MAX_DEPTH];
            integer currentChildNumber[MAX_DEPTH];

            __shared__ volatile real cellSize[MAX_DEPTH];

            if (threadIdx.x == 0) {
                cellSize[0] = 4.0 * radius * radius;
#pragma unroll
                for (i = 1; i < MAX_DEPTH; i++) {
                    cellSize[i] = cellSize[i - 1] * 0.25;
                }
            }

            __syncthreads();

            for (i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += blockDim.x * gridDim.x) {

                px = particles->x[i];
#if DIM > 1
                py = particles->y[i];
#if DIM == 3
                pz = particles->z[i];
#endif
#endif
                //particles->ax[i] = 0.0;
                particles->g_ax[i] = 0.0;
#if DIM > 1
                //particles->ay[i] = 0.0;
                particles->g_ay[i] = 0.0;
#if DIM == 3
                //particles->az[i] = 0.0;
                particles->g_az[i] = 0.0;
#endif
#endif
                ax = 0.0;
#if DIM > 1
                ay = 0.0;
#if DIM == 3
                az = 0.0;
#endif
#endif

                // start at root
                depth = 1;
                currentNodeIndex[depth] = 0;
                currentChildNumber[depth] = 0;

                do {
                    childNumber = currentChildNumber[depth];
                    nodeIndex = currentNodeIndex[depth];

                    while(childNumber < POW_DIM) {
                        do {
                            child = tree->child[POW_DIM * nodeIndex + childNumber]; //childList[childListIndex(nodeIndex, childNumber)];
                            childNumber++;
                        } while(child == -1 && childNumber < POW_DIM);
                        if (child != -1 && child != i) { // dont do self-gravity with yourself!
                            dx = particles->x[child] - px;
                            distance = dx*dx + smoothing; //150329404.287723; //(0.0317 * 0.0317); //0.025;
#if DIM > 1
                            dy = particles->y[child] - py;
                            distance += dy*dy;
#endif
#if DIM == 3
                            dz = particles->z[child] - pz;
                            distance += dz*dz;
#endif
                            // if child is leaf or far away
                            if (child < n || distance * thetasq > cellSize[depth]) {
                                distance = sqrt(distance);
#if SI_UNITS
                                f = Constants::G * particles->mass[child] / (distance * distance * distance);
#else
                                f = particles->mass[child] / (distance * distance * distance);
#endif

                                ax += f*dx;
#if DIM > 1
                                ay += f*dy;
#if DIM == 3
                                az += f*dz;
#endif
#endif
                                //TODO: some flag for calculating potential energy
                                // gravitational potential energy
                                //particles->u[i] -= 0.5 * (particles->mass[child] * particles->mass[i])/distance;
                                // end: gravitational potential energy
                            } else {
                                // put child on stack
                                currentChildNumber[depth] = childNumber;
                                currentNodeIndex[depth] = nodeIndex;
                                depth++;
                                if (depth == MAX_DEPTH) {
                                    cudaTerminate("depth = %i >= MAX_DEPTH = %i\n", depth, MAX_DEPTH);
                                }
                                childNumber = 0;
                                nodeIndex = child;
                            }
                        }
                    }
                    depth--;
                } while(depth > 0);

                //particles->ax[i] = ax;
                particles->g_ax[i] = ax;
#if DIM > 1
                //particles->ay[i] = ay;
                particles->g_ay[i] = ay;
#if DIM == 3
                //particles->az[i] = az;
                particles->g_az[i] = az;
#endif
#endif
            }
        }

        __global__ void computeForces_v1_2(Tree *tree, Particles *particles, real radius, integer n, integer m,
                                           SubDomainKeyTree *subDomainKeyTree, real theta, real smoothing) {

            register int i, ii;
            int child, nodeIndex, childNumber, depth;

            real px, ax, dx, f, distance;
#if DIM > 1
            real py, ay, dy;
#if DIM == 3
            real pz, az, dz;
#endif
#endif
            real sml;
            real thetasq = theta*theta;

            __shared__ int currentNodeIndex[MAX_DEPTH];
            __shared__ int currentChildNumber[MAX_DEPTH];

            __shared__ volatile real cellSize[MAX_DEPTH];

            if (threadIdx.x == 0) {
                cellSize[0] = 4.0 * radius * radius;
                #pragma unroll
                for (i = 1; i < MAX_DEPTH; i++) {
                    cellSize[i] = cellSize[i - 1] * 0.25;
                }
            }

            __syncthreads();

            for (ii = threadIdx.x + blockIdx.x * blockDim.x; ii < n; ii += blockDim.x * gridDim.x) {

                i = tree->sorted[ii]; //i = ii;

                px = particles->x[i];
#if DIM > 1
                py = particles->y[i];
#if DIM == 3
                pz = particles->z[i];
#endif
#endif
                //particles->ax[i] = 0.0;
                particles->g_ax[i] = 0.0;
#if DIM > 1
                //particles->ay[i] = 0.0;
                particles->g_ay[i] = 0.0;
#if DIM == 3
                //particles->az[i] = 0.0;
                particles->g_az[i] = 0.0;
#endif
#endif
                ax = 0.0;
#if DIM > 1
                ay = 0.0;
#if DIM == 3
                az = 0.0;
#endif
#endif

                // start at root
                depth = 1;
                currentNodeIndex[depth] = 0;
                currentChildNumber[depth] = 0;

                do {
                    childNumber = currentChildNumber[depth];
                    nodeIndex = currentNodeIndex[depth];

                    while(childNumber < POW_DIM) {
                        do {
                            child = tree->child[POW_DIM * nodeIndex + childNumber];
                            childNumber++;
                        } while(child == -1 && childNumber < POW_DIM);
                        if (child != -1 && child != i) { // dont do self-gravity with yourself!
                            dx = particles->x[child] - px;
                            distance = dx*dx + smoothing; //150329404.287723; //(0.0317 * 0.0317); //0.025;
#if DIM > 1
                            dy = particles->y[child] - py;
                            distance += dy*dy;
#endif
#if DIM == 3
                            dz = particles->z[child] - pz;
                            distance += dz*dz;
#endif
                            // if child is leaf or far away
                            if (child < n || distance * thetasq > cellSize[depth]) {
                                distance = cuda::math::sqrt(distance);
#if SI_UNITS
                                f = Constants::G * particles->mass[child] / (distance * distance * distance);
#else
                                f = particles->mass[child] / (distance * distance * distance);
#endif

                                ax += f*dx;
#if DIM > 1
                                ay += f*dy;
#if DIM == 3
                                az += f*dz;
#endif
#endif
                                //TODO: some flag for calculating potential energy
                                // gravitational potential energy
                                //particles->u[i] -= 0.5 * (particles->mass[child] * particles->mass[i])/distance;
                                // end: gravitational potential energy
                            } else {
                                // put child on stack
                                currentChildNumber[depth] = childNumber;
                                currentNodeIndex[depth] = nodeIndex;
                                depth++;
                                if (depth == MAX_DEPTH) {
                                    cudaTerminate("depth = %i >= MAX_DEPTH = %i\n", depth, MAX_DEPTH);
                                }
                                childNumber = 0;
                                nodeIndex = child;
                            }
                        }
                    }
                    depth--;
                } while(depth > 0);

                //particles->ax[i] = ax;
                particles->g_ax[i] = ax;
#if DIM > 1
                //particles->ay[i] = ay;
                particles->g_ay[i] = ay;
#if DIM == 3
                //particles->az[i] = az;
                particles->g_az[i] = az;
#endif
#endif
            }
        }

        // see: https://iss.oden.utexas.edu/Publications/Papers/burtscher11.pdf
        /*__global__ void test() {
            // precompute and cache info
            // determine first thread in each warp
            for (//sorted body indexes assigned to me) {
                // cache body data
                // initialize iteration stack
                depth = 0;
                while (depth >= 0) {
                    while (//there are more nodes to visit) {
                        if (//I’m the first thread in the warp) {
                            // move on to next node
                            // read node data and put in shared memory
                        }
                        threadfence block();
                        if (//node is not null) {
                            // get node data from shared memory
                            // compute distance to node
                            if ((//node is a body) || all(//distance >= cutoff)) {
                                // compute interaction force contribution
                            } else {
                                depth++; // descend to next tree level
                                if (//I’m the first thread in the warp) {
                                    // push node’s children onto iteration stack
                                }
                                threadfence block();
                            }
                        } else {
                            depth = max(0, depth-1); // early out because remaining nodes are also null
                        }
                    }
                    depth−−;
                }
            // update body data
            }
        }*/

        __global__ void computeForces_v2(Tree *tree, Particles *particles, real radius, integer n, integer m,
                                         integer blockSize, integer warp, integer stackSize,
                                         SubDomainKeyTree *subDomainKeyTree, real theta,
                                         real smoothing) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;

            register int sortedIndex;

            //__shared__ real depth[stackSize * blockSize/warp];
            //__shared__ integer stack[stackSize * blockSize/warp];
            extern __shared__ real buffer[];

            real* depth = (real*)buffer;
            integer* stack = (integer*)&depth[stackSize * blockSize/warp];

            real pos_x;
#if DIM > 1
            real pos_y;
#if DIM == 3
            real pos_z;
#endif
#endif

            real acc_x;
#if DIM > 1
            real acc_y;
#if DIM == 3
            real acc_z;
#endif
#endif

            // in case that one of the first children are a leaf
            int jj = -1;
            #pragma unroll
            for (integer i=0; i<POW_DIM; i++) {
                if (tree->child[i] != -1) {
                    jj++;
                }
            }

            int counter = threadIdx.x % warp;
            int stackStartIndex = stackSize*(threadIdx.x / warp);

            while ((bodyIndex + offset) < n) {

                //sortedIndex = bodyIndex + offset;
                sortedIndex = tree->sorted[bodyIndex + offset];

                pos_x = particles->x[sortedIndex];
#if DIM > 1
                pos_y = particles->y[sortedIndex];
#if DIM == 3
                pos_z = particles->z[sortedIndex];
#endif
#endif

                acc_x = 0.0;
#if DIM > 1
                acc_y = 0.0;
#if DIM == 3
                acc_z = 0.0;
#endif
#endif

                // initialize stack
                integer top = jj + stackStartIndex;

                if (counter == 0) {

                    int temp = 0;

                    #pragma unroll
                    for (int i=0; i<POW_DIM; i++) {
                        if (tree->child[i] != -1) {
                            stack[stackStartIndex + temp] = tree->child[i];
                            depth[stackStartIndex + temp] = radius*radius/theta;
                            temp++;
                        }
                    }
                }
                __syncthreads();

                // while stack is not empty / more nodes to visit
                while (top >= stackStartIndex) {

                    integer node = stack[top];

                    real dp = 0.25 * depth[top]; //powf(0.5, DIM) * depth[top]; //0.25*depth[top]; // float dp = depth[top];

                    for (integer i=0; i<POW_DIM; i++) {

                        integer ch = tree->child[POW_DIM*node + i];

                        //__threadfence();

                        if (ch >= 0) {

                            real dx = particles->x[ch] - pos_x;
#if DIM > 1
                            real dy = particles->y[ch] - pos_y;
#if DIM == 3
                            real dz = particles->z[ch] - pos_z;
#endif
#endif

                            real r = dx*dx + smoothing; // SMOOTHING
#if DIM > 1
                            r += dy*dy;
#if DIM == 3
                            r += dz*dz;
#endif
#endif

                            //if (ch < n /*is leaf node*/ || !__any_sync(activeMask, dp > r)) {
                            if (ch < m /*is leaf node*/ || __all_sync(__activemask(), dp <= r)) {

                                // calculate interaction force contribution
                                if (r > 0.f) { //NEW //TODO: how to avoid r = 0?
                                    r = cuda::math::rsqrt(r);
                                }

#if SI_UNITS
                                real f = Constants::G * particles->mass[ch] * r * r * r;
#else
                                real f = particles->mass[ch] * r * r * r;
#endif

                                acc_x += f*dx;
#if DIM > 1
                                acc_y += f*dy;
#if DIM == 3
                                acc_z += f*dz;
#endif
#endif
                            }
                            else {
                                // if first thread in warp: push node's children onto iteration stack
                                if (counter == 0) {
                                    stack[top] = ch;
                                    depth[top] = dp; // depth[top] = 0.25*dp;
                                }
                                top++; // descend to next tree level
                                __threadfence_block();
                            }
                        }
                        // this is not working
                        //else {
                        //    top = cuda::math::max(stackStartIndex, top-1);
                        //}
                    }
                    top--;
                }
                // update body data
                particles->g_ax[sortedIndex] = acc_x;
#if DIM > 1
                particles->g_ay[sortedIndex] = acc_y;
#if DIM == 3
                particles->g_az[sortedIndex] = acc_z;
#endif
#endif

                offset += stride;
                __syncthreads();
            }

        }

        __global__ void computeForces_v2_1(Tree *tree, Particles *particles, integer n, integer m, integer blockSize,
                                           integer warp, integer stackSize, SubDomainKeyTree *subDomainKeyTree,
                                           real theta, real smoothing) {

            integer bodyIndex = threadIdx.x + blockIdx.x*blockDim.x;
            integer stride = blockDim.x*gridDim.x;
            integer offset = 0;

            //__shared__ real depth[stackSize * blockSize/warp];
            //__shared__ integer stack[stackSize * blockSize/warp];
            extern __shared__ real buffer[];

            real* depth = (real*)buffer;
            integer* stack = (integer*)&depth[stackSize * blockSize/warp];

            real x_radius = 0.5*(*tree->maxX - (*tree->minX));
#if DIM > 1
            real y_radius = 0.5*(*tree->maxY - (*tree->minY));
#if DIM == 3
            real z_radius = 0.5*(*tree->maxZ - (*tree->minZ));
#endif
#endif

#if DIM == 1
            real radius = x_radius;
#elif DIM == 2
            real radius = cuda::math::max(x_radius, y_radius);
#else
            real radius_max = cuda::math::max(x_radius, y_radius);
            real radius = cuda::math::max(radius_max, z_radius);
#endif

            // in case that one of the first children are a leaf
            integer jj = -1;
            for (integer i=0; i<POW_DIM; i++) {
                if (tree->child[i] != -1) {
                    jj++;
                }
            }

            integer counter = threadIdx.x % warp;
            integer stackStartIndex = stackSize*(threadIdx.x / warp);

            while ((bodyIndex + offset) < n) {

                real pos_x = particles->x[bodyIndex + offset];
#if DIM > 1
                real pos_y = particles->y[bodyIndex + offset];
#if DIM == 3
                real pos_z = particles->z[bodyIndex + offset];
#endif
#endif

                real acc_x = 0.0;
#if DIM > 1
                real acc_y = 0.0;
#if DIM == 3
                real acc_z = 0.0;
#endif
#endif

                // initialize stack
                integer top = jj + stackStartIndex;

                if (counter == 0) {

                    integer temp = 0;

                    for (int i=0; i<POW_DIM; i++) {
                        if (tree->child[i] != -1) {
                            stack[stackStartIndex + temp] = tree->child[i];
                            depth[stackStartIndex + temp] = radius*radius/theta;
                            temp++;
                        }
                    }
                }
                __syncthreads();

                // while stack is not empty / more nodes to visit
                while (top >= stackStartIndex) {

                    integer node = stack[top];

                    real dp = 0.5 * depth[top]; //powf(0.5, DIM) * depth[top]; //0.25*depth[top]; // float dp = depth[top];

                    for (integer i=0; i<POW_DIM; i++) {

                        integer ch = tree->child[POW_DIM*node + i];

                        //__threadfence();

                        if (ch >= 0) {

                            real dx = particles->x[ch] - pos_x;
#if DIM > 1
                            real dy = particles->y[ch] - pos_y;
#if DIM == 3
                            real dz = particles->z[ch] - pos_z;
#endif
#endif

                            real r = dx*dx + smoothing; // SMOOTHING
#if DIM > 1
                            r += dy*dy;
#if DIM == 3
                            r += dz*dz;
#endif
#endif

                            //if (ch < n /*is leaf node*/ || !__any_sync(activeMask, dp > r)) {
                            if (ch < m /*is leaf node*/ || __all_sync(__activemask(), dp <= r)) {

                                // calculate interaction force contribution
                                if (r > 0.f) { //NEW //TODO: how to avoid r = 0?
                                    r = cuda::math::rsqrt(r);
                                }

#if SI_UNITS
                                real f = Constants::G * particles->mass[ch] * r * r * r;
#else
                                real f = particles->mass[ch] * r * r * r;
#endif

                                acc_x += f*dx;
#if DIM > 1
                                acc_y += f*dy;
#if DIM == 3
                                acc_z += f*dz;
#endif
#endif
                            }
                            else {
                                // if first thread in warp: push node's children onto iteration stack
                                if (counter == 0) {
                                    stack[top] = ch;
                                    depth[top] = dp; // depth[top] = 0.25*dp;
                                }
                                top++; // descend to next tree level
                                //__threadfence();
                            }
                        }
                        else {
                            /*top = max(stackStartIndex, top-1); */
                        }
                    }
                    top--;
                }
                // update body data
                particles->g_ax[bodyIndex + offset] = acc_x;
#if DIM > 1
                particles->g_ay[bodyIndex + offset] = acc_y;
#if DIM == 3
                particles->g_az[bodyIndex + offset] = acc_z;
#endif
#endif
                offset += stride;

                __syncthreads();
            }

        }

        __global__ void createKeyHistRanges(Helper *helper, integer bins) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            keyType max_key = 1UL << (DIM * 21);//1UL << 63;

            while ((bodyIndex + offset) < bins) {

                helper->keyTypeBuffer[bodyIndex + offset] = (bodyIndex + offset) * (max_key/bins);
                //printf("keyHistRanges[%i] = %lu\n", bodyIndex + offset, keyHistRanges[bodyIndex + offset]);

                if ((bodyIndex + offset) == (bins - 1)) {
                    helper->keyTypeBuffer[bins-1] = KEY_MAX;
                }
                offset += stride;
            }
        }

        __global__ void intermediateSymbolicForce(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                                  DomainList *domainList, integer *sendIndices, real diam, real theta_,
                                                  integer n, integer m, integer relevantIndex, integer level,
                                                  Curve::Type curveType) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            while ((bodyIndex + offset) < *tree->index) {
                if (sendIndices[bodyIndex + offset] == 2) {
                    sendIndices[bodyIndex + offset] = 0;
                }
                if (sendIndices[bodyIndex + offset] == 3) {
                    sendIndices[bodyIndex + offset] = 1;
                }

                offset += stride;
            }
        }

        __global__ void symbolicForce(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                      DomainList *domainList, integer *sendIndices, real diam, real theta_,
                                      integer n, integer m, integer relevantIndex, integer level,
                                      Curve::Type curveType) {

            int bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            int stride = blockDim.x * gridDim.x;
            int offset = 0;

            int particleLevel;
            int domainListLevel;
            int currentDomainListIndex;
            int currentParticleIndex;
            int childIndex;

            int childPath;
            int tempChildIndex;

            bool isDomainListNode;
            bool insert;

            real min_x, max_x;
            real dx;
#if DIM > 1
            real min_y, max_y;
            real dy;
#if DIM == 3
            real min_z, max_z;
            real dz;
#endif
#endif
            real r;

            // IDEA: sendIndices = [-1, -1, -1, ..., -1, -1]
            // mark to be tested indices with 2's: e.g.: sendIndices = [-1, 2, -1, ..., 2, -1]
            // the 2's are converted within a separate kernel to zeros (which will be tested within this kernel)
            //  separate kernel necessary to avoid race conditions
            // mark to be sent indices/particles with 3's: e.g.: sendIndices = [-1, 0, 3, ..., 3, 3]
            //  the 3's are converted within a separate kernel to ones

            if (level == 0) { // mark first level children as starting point
                while ((bodyIndex + offset) < POW_DIM) {
                    //if ((bodyIndex + offset) == 0) {
                    //    printf("symbolicForce: [rank %i] relevantDomainListIndices[%i] = %i (%f, %f, %f)\n",
                    //           subDomainKeyTree->rank,
                    //           relevantIndex, domainList->relevantDomainListIndices[relevantIndex],
                    //           particles->x[domainList->relevantDomainListIndices[relevantIndex]],
                    //           particles->y[domainList->relevantDomainListIndices[relevantIndex]],
                    //           particles->z[domainList->relevantDomainListIndices[relevantIndex]]);
                    //}
                    if (tree->child[bodyIndex + offset] != -1) {
                        sendIndices[tree->child[bodyIndex + offset]] = 0;
                    }
                    offset += stride;
                }
            }
            else {

                while ((bodyIndex + offset) < *tree->index) {

                    //if (bodyIndex + offset == 0) {
                    //    printf("[rank %i] relevantIndex = %i domainListIndex = %i (%f, %f, %f) %f\n", subDomainKeyTree->rank,
                    //           relevantIndex, domainList->relevantDomainListIndices[relevantIndex],
                    //           particles->x[domainList->relevantDomainListIndices[relevantIndex]],
                    //           particles->y[domainList->relevantDomainListIndices[relevantIndex]],
                    //           particles->z[domainList->relevantDomainListIndices[relevantIndex]],
                    //           particles->mass[domainList->relevantDomainListIndices[relevantIndex]]);
                    //}

                    currentParticleIndex = bodyIndex + offset;

                    if ((sendIndices[currentParticleIndex] == 0 || sendIndices[currentParticleIndex] == 3) && (currentParticleIndex < n || currentParticleIndex >= m )) {

                        insert = true;
                        isDomainListNode = false;

                        if (sendIndices[currentParticleIndex] == 0) {

                            // check whether to be inserted index corresponds to a domain list
                            //if (insert) {
                                //for (int i_domain = 0; i_domain < *domainList->domainListIndex; i_domain++) {
                                //    if ((bodyIndex + offset) == domainList->domainListIndices[i_domain]) {
                                //        insert = false;
                                //        isDomainListNode = true;
                                //        break;
                                //    }
                                //}
                            if (particles->nodeType[bodyIndex + offset] >= 1) {
                                insert = false;
                                isDomainListNode = true;
                            }
                            //}
                            // TODO: this is probably not necessary, since only domain list indices can correspond to another process
                            if (!isDomainListNode) {
                                if (subDomainKeyTree->key2proc(
                                        tree->getParticleKey(particles, currentParticleIndex, MAX_LEVEL, curveType)) !=
                                    subDomainKeyTree->rank) {
                                    insert = false;
                                    //printf("Happening?\n");
                                }
                            }

                            if (insert) {
                                sendIndices[currentParticleIndex] = 3;
                            } else {
                                sendIndices[currentParticleIndex] = -1;
                            }
                        }

                        // get the particle's level
                        //particleLevel /*int tempParticleLevel*/ = tree->getTreeLevel(particles, currentParticleIndex, MAX_LEVEL, curveType);
                        particleLevel = particles->level[currentParticleIndex];
                        //if (tempParticleLevel != particleLevel) {
                        //    printf("%i vs %i\n", tempParticleLevel, particleLevel);
                        //}
#if DEBUGGING
#if DIM == 3
                        if (particleLevel < 0) {
                            printf("WTF particleLevel = %i for %i (%e, %e, %e) (numParticlesLocal = %i, index = %i)\n",
                                   particleLevel, currentParticleIndex, particles->x[currentParticleIndex],
                                   particles->y[currentParticleIndex], particles->z[currentParticleIndex],
                                   n, *tree->index);
                        }
#endif
#endif

                        // get the domain list node's level
                        //domainListLevel = tree->getTreeLevel(particles,
                        //                                     domainList->relevantDomainListIndices[relevantIndex],
                        //                                     MAX_LEVEL, curveType);
                        domainListLevel = domainList->relevantDomainListLevels[relevantIndex];
                        currentDomainListIndex = domainList->relevantDomainListIndices[relevantIndex];
                        //printf("domainListLevel = %i\n", domainListLevel);
                        if (domainListLevel == -1) {
                            cudaAssert("symbolicForce(): domainListLevel == -1 for (relevant) index: %i\n",
                                       relevantIndex);
                        }

                        min_x = *tree->minX;
                        max_x = *tree->maxX;
#if DIM > 1
                        min_y = *tree->minY;
                        max_y = *tree->maxY;
#if DIM == 3
                        min_z = *tree->minZ;
                        max_z = *tree->maxZ;
#endif
#endif

                        // determine domain list node's bounding box (in order to determine the distance)
                        //if (domainListLevel != 1) {
                        //    printf("domainListLevel = %i\n", domainListLevel);
                        //    assert(0);
                        //}
                        for (int j = 0; j < domainListLevel; j++) {

                            /*
#if DIM == 3
                            if (particles->x[domainList->relevantDomainListIndices[relevantIndex]] <= max_x && particles->x[domainList->relevantDomainListIndices[relevantIndex]] >= min_x &&
                                particles->y[domainList->relevantDomainListIndices[relevantIndex]] <= max_y && particles->y[domainList->relevantDomainListIndices[relevantIndex]] >= min_y &&
                                particles->z[domainList->relevantDomainListIndices[relevantIndex]] <= max_z && particles->z[domainList->relevantDomainListIndices[relevantIndex]] >= min_z) {

                            }
                            else {
                                printf("not within box %i, %i  level: %i (%f, %f, %f) box (%f, %f), (%f, %f), (%f, %f)!\n", relevantIndex, domainList->relevantDomainListIndices[relevantIndex],
                                       domainList->relevantDomainListLevels[relevantIndex],
                                       particles->x[domainList->relevantDomainListIndices[relevantIndex]], particles->y[domainList->relevantDomainListIndices[relevantIndex]],
                                       particles->z[domainList->relevantDomainListIndices[relevantIndex]],
                                       min_x, max_x, min_y, max_y, min_z, max_z);
                                assert(0);
                            }
#endif
                            */

                            childPath = 0;
                            if (particles->x[currentDomainListIndex] < 0.5 * (min_x + max_x)) {
                                childPath += 1;
                                max_x = 0.5 * (min_x + max_x);
                            } else {
                                min_x = 0.5 * (min_x + max_x);
                            }
#if DIM > 1
                            if (particles->y[currentDomainListIndex] < 0.5 * (min_y + max_y)) {
                                childPath += 2;
                                max_y = 0.5 * (min_y + max_y);
                            } else {
                                min_y = 0.5 * (min_y + max_y);
                            }
#if DIM == 3
                            if (particles->z[currentDomainListIndex] < 0.5 * (min_z + max_z)) {
                                childPath += 4;
                                max_z = 0.5 * (min_z + max_z);
                            } else {
                                min_z = 0.5 * (min_z + max_z);
                            }
#endif
#endif
                        }

                        // determine (smallest) distance between domain list box and (pseudo-) particle
                        if (particles->x[currentParticleIndex] < min_x) {
                            dx = particles->x[currentParticleIndex] - min_x;
                        } else if (particles->x[currentParticleIndex] > max_x) {
                            dx = particles->x[currentParticleIndex] - max_x;
                        } else {
                            dx = 0.;
                        }
#if DIM > 1
                        if (particles->y[currentParticleIndex] < min_y) {
                            dy = particles->y[currentParticleIndex] - min_y;
                        } else if (particles->y[currentParticleIndex] > max_y) {
                            dy = particles->y[currentParticleIndex] - max_y;
                        } else {
                            dy = 0.;
                        }
#if DIM == 3
                        if (particles->z[currentParticleIndex] < min_z) {
                            dz = particles->z[currentParticleIndex] - min_z;
                        } else if (particles->z[currentParticleIndex] > max_z) { dz =
                        particles->z[currentParticleIndex] - max_z;
                        } else {
                            dz = 0.;
                        }

#endif
#endif

#if DIM == 1
                        r = cuda::math::sqrt(dx*dx);
#elif DIM == 2
                        r = cuda::math::sqrt(dx*dx + dy*dy);
#else
                        r = cuda::math::sqrt(dx*dx + dy*dy + dz*dz);
#endif

                        // TODO: depending on gravity force version and amount of processes: 2 * diam or 1 * diam (why?)
                        //printf("%f >= %f (particleLevel = %i, theta = %f, r = %f)\n", powf(0.5, particleLevel-1) /* * 2*/ * diam, (theta_ * r), particleLevel, theta_, r);
                        if (particleLevel != -1 && (((powf(0.5, particleLevel-1) /* * 2*/ * diam) >= (theta_ * r)) || isDomainListNode)) {

                            #pragma unroll
                            for (int i = 0; i < POW_DIM; i++) {

                                //if (sendIndices[tree->child[POW_DIM * (bodyIndex + offset) + i]] != 1 && tree->child[POW_DIM * (bodyIndex + offset) + i] != -1) {
                                //    sendIndices[tree->child[POW_DIM * (bodyIndex + offset) + i]] = 2;
                                //}

                                //if (insert && tree->child[POW_DIM * (bodyIndex + offset) + i] != -1 && particles->x[tree->child[POW_DIM * (bodyIndex + offset) + i]] == particles->x[bodyIndex + offset] &&
                                //        particles->y[tree->child[POW_DIM * (bodyIndex + offset) + i]] == particles->y[bodyIndex + offset]) {
                                    //printf("[rank %i] index = %i == child = %i ^= %i (%f, %f, %f) vs (%f, %f, %f)\n", subDomainKeyTree->rank, bodyIndex + offset, i, tree->child[POW_DIM * (bodyIndex + offset) + i],
                                    //       particles->x[bodyIndex + offset], particles->y[bodyIndex + offset], particles->z[bodyIndex + offset],
                                    //
                                    //       particles->x[tree->child[POW_DIM * (bodyIndex + offset) + i]], particles->y[tree->child[POW_DIM * (bodyIndex + offset) + i]], particles->z[tree->child[POW_DIM * (bodyIndex + offset) + i]]);
                                //}

                                if (tree->child[POW_DIM * currentParticleIndex + i] != -1) {
                                    if (sendIndices[tree->child[POW_DIM * currentParticleIndex + i]] != 1) {
                                        sendIndices[tree->child[POW_DIM * currentParticleIndex + i]] = 2;
                                    }
                                }

                            }
                        }
                    }

                    __threadfence();
                    offset += stride;
                }

            }

        }

        __global__ void compTheta(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                  DomainList *domainList, Helper *helper, Curve::Type curveType) {

            integer index = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;
            integer bodyIndex;
            keyType key, hilbert;
            integer domainIndex;
            integer proc;

            //"loop" over domain list nodes
            while ((index + offset) < *domainList->domainListIndex) {

                bodyIndex = domainList->domainListIndices[index + offset];
                //calculate key
                //TODO: why not
                //key =  domainList->domainListKeys[index + offset]; //???
                //hilbert = KeyNS::lebesgue2hilbert(key, 21);
                key = tree->getParticleKey(particles, bodyIndex, MAX_LEVEL, curveType); // working version
                //if domain list node belongs to other process: add to relevant domain list indices
                proc = subDomainKeyTree->key2proc(key);

                //printf("[rank %i] potential relevant domain list node: %i (%f, %f, %f)\n", subDomainKeyTree->rank,
                //       bodyIndex, particles->x[bodyIndex],
                //       particles->y[bodyIndex], particles->z[bodyIndex]);

                if (proc < 0 && particles->mass[bodyIndex] <= 0.f) {
                    printf("proc = %i, mass = %e\n", proc, particles->mass[bodyIndex]);
                    //assert(0);
                }
                if (proc != subDomainKeyTree->rank && proc >= 0 && particles->mass[bodyIndex] > 0.f) {
                    //printf("[rank = %i] proc = %i, key = %lu for x = (%f, %f, %f)\n", subDomainKeyTree->rank, proc, key, particles->x[bodyIndex], particles->y[bodyIndex], particles->z[bodyIndex]);
                    domainIndex = atomicAdd(domainList->domainListCounter, 1);
                    domainList->relevantDomainListIndices[domainIndex] = bodyIndex;
                    domainList->relevantDomainListLevels[domainIndex] = domainList->domainListLevels[index + offset];
                    domainList->relevantDomainListProcess[domainIndex] = proc;

                    //printf("[rank %i] Adding relevant domain list node: %i (%f, %f, %f)\n", subDomainKeyTree->rank,
                    //       bodyIndex, particles->x[bodyIndex],
                    //       particles->y[bodyIndex], particles->z[bodyIndex]);
                }
                offset += stride;
            }

        }

        __global__ void keyHistCounter(Tree *tree, Particles *particles, SubDomainKeyTree *subDomainKeyTree,
                                       Helper *helper,
                                       /*keyType *keyHistRanges, integer *keyHistCounts,*/ int bins, int n,
                                       Curve::Type curveType) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            keyType key;

            while ((bodyIndex + offset) < n) {

                key = tree->getParticleKey(particles, bodyIndex + offset, MAX_LEVEL, curveType);

                for (int i = 0; i < (bins); i++) {
                    if (key >= helper->keyTypeBuffer[i] && key < helper->keyTypeBuffer[i + 1]) {
                        //keyHistCounts[i] += 1;
                        atomicAdd(&helper->integerBuffer[i], 1);
                        break;
                    }
                }

                offset += stride;
            }

        }

        //TODO: resetting helper (buffers)?!
        __global__ void calculateNewRange(SubDomainKeyTree *subDomainKeyTree, Helper *helper,
                                          /*keyType *keyHistRanges, integer *keyHistCounts,*/ int bins, int n,
                                          Curve::Type curveType) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            integer sum;
            keyType newRange;

            while ((bodyIndex + offset) < (bins-1)) {

                sum = 0;
                for (integer i=0; i<(bodyIndex+offset); i++) {
                    sum += helper->integerBuffer[i];
                }

                for (integer i=1; i<subDomainKeyTree->numProcesses; i++) {
                    if ((sum + helper->integerBuffer[bodyIndex + offset]) >= (i*n) && sum < (i*n)) {
                        printf("[rank %i] new range: %lu\n", subDomainKeyTree->rank,
                               helper->keyTypeBuffer[bodyIndex + offset]);
                        subDomainKeyTree->range[i] = (helper->keyTypeBuffer[bodyIndex + offset] >> (1*DIM)) << (1*DIM);
                    }
                }


                //printf("[rank %i] keyHistCounts[%i] = %i\n", s->rank, bodyIndex+offset, keyHistCounts[bodyIndex+offset]);
                atomicAdd(helper->integerVal, helper->integerBuffer[bodyIndex+offset]);
                offset += stride;
            }

        }

        __global__ void insertReceivedPseudoParticles(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                                      integer *levels, int level, int n, int m) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset;

            integer childPath;
            integer temp;

            integer insertionLevel;

            real min_x, max_x;
#if DIM > 1
            real min_y, max_y;
#if DIM == 3
            real min_z, max_z;
#endif
#endif

            // debug
            //if (bodyIndex == 0 && level == 0) {
            //    integer levelCounter;
            //    for (int debugLevel = 0; debugLevel< MAX_LEVEL; debugLevel++) {
            //        levelCounter = 0;
            //        for (int i = 0; i < (tree->toDeleteNode[1] - tree->toDeleteNode[0]); i++) {
            //            if (debugLevel == 0) {
            //                if (subDomainKeyTree->key2proc(tree->getParticleKey(particles, tree->toDeleteNode[0] + i, MAX_LEVEL, Curve::lebesgue)) == subDomainKeyTree->rank) {
            //                    printf("\n-------------------------------------------------\nATTENTION\n\n-------------------------------------------------\n");
            //                }
            //                //if (particles->x[tree->toDeleteNode[0] + i] == 0.f && particles->y[tree->toDeleteNode[0] + i] == 0.f &&
            //                //        particles->z[tree->toDeleteNode[0] + i] == 0.f) {
            //                //    printf("\n-------------------------------------------------\nATTENTION\n\n-------------------------------------------------\n");
            //                //}
            //              //printf("[rank %i] index = %i level = %i x = (%f, %f, %f) m = %f\n", subDomainKeyTree->rank,
            //                //       tree->toDeleteNode[0] + i,
            //                //       levels[i],
            //                //       particles->x[tree->toDeleteNode[0] + i],
            //                //       particles->y[tree->toDeleteNode[0] + i],
            //                //       particles->z[tree->toDeleteNode[0] + i],
            //                //       particles->mass[tree->toDeleteNode[0] + i]);
            //            }
            //            if (levels[i] == debugLevel) {
            //                //printf("[rank %i] level available: %i\n", subDomainKeyTree->rank, debugLevel);
            //                levelCounter++;
            //            }
            //        }
            //        if (levelCounter > 0) {
            //            printf("[rank %i] level available: %i (# = %i)\n", subDomainKeyTree->rank, debugLevel, levelCounter);
            //        }
            //    }
            //}

            offset = 0;
            while ((bodyIndex + offset) < (tree->toDeleteNode[1] - tree->toDeleteNode[0])) {

                insertionLevel = 0;

                //if (levels[bodyIndex + offset] < 0 || levels[bodyIndex + offset] > 21) {
                //    printf("[rank %i] levels[%i] = %i!\n", subDomainKeyTree->rank, bodyIndex + offset, levels[bodyIndex + offset]);
                //    assert(0);
                //}

                if (levels[bodyIndex + offset] == level) {

                    min_x = *tree->minX;
                    max_x = *tree->maxX;
#if DIM > 1
                    min_y = *tree->minY;
                    max_y = *tree->maxY;
#if DIM == 3
                    min_z = *tree->minZ;
                    max_z = *tree->maxZ;
#endif
#endif
                    temp = 0;
                    childPath = 0;

                    // find insertion point for body
                    if (particles->x[tree->toDeleteNode[0] + bodyIndex + offset] < 0.5 * (min_x + max_x)) { // x direction
                        childPath += 1;
                        max_x = 0.5 * (min_x + max_x);
                    }
                    else {
                        min_x = 0.5 * (min_x + max_x);
                    }
#if DIM > 1
                    if (particles->y[tree->toDeleteNode[0] + bodyIndex + offset] < 0.5 * (min_y + max_y)) { // y direction
                        childPath += 2;
                        max_y = 0.5 * (min_y + max_y);
                    }
                    else {
                        min_y = 0.5 * (min_y + max_y);
                    }
#if DIM == 3
                    if (particles->z[tree->toDeleteNode[0] + bodyIndex + offset] < 0.5 * (min_z + max_z)) {  // z direction
                        childPath += 4;
                        max_z = 0.5 * (min_z + max_z);
                    }
                    else {
                        min_z = 0.5 * (min_z + max_z);
                    }
#endif
#endif
                    int childIndex = tree->child[temp*POW_DIM + childPath];
                    atomicAdd(&tree->count[childIndex], 1);
                    insertionLevel++;

                    // debug
                    //if (subDomainKeyTree->rank == 0) {
                    //    if (childPath < 4) {
                    //        printf("[rank %i] childPath = %i WTF?\n", subDomainKeyTree->rank, childPath);
                    //    }
                    //}
                    //else {
                    //    if (childPath >= 4) {
                    //        printf("[rank %i] childPath = %i WTF?\n", subDomainKeyTree->rank, childPath);
                    //    }
                    //}
                    // end: debug

                    // debug
                    //if ((bodyIndex + offset) % 100 == 0) {
                    //    printf("[rank %i] childPath = %i, childIndex = %i\n", subDomainKeyTree->rank, childPath,
                    //           childIndex);
                    //}
                    // end: debug

                    // traverse tree until hitting leaf node
                    while (childIndex >= m) {
                        insertionLevel++;

                        temp = childIndex;
                        childPath = 0;

                        // find insertion point for body
                        if (particles->x[tree->toDeleteNode[0] + bodyIndex + offset] < 0.5 * (min_x + max_x)) { // x direction
                            childPath += 1;
                            max_x = 0.5 * (min_x + max_x);
                        }
                        else {
                            min_x = 0.5 * (min_x + max_x);
                        }
#if DIM > 1
                        if (particles->y[tree->toDeleteNode[0] + bodyIndex + offset] < 0.5 * (min_y + max_y)) { // y direction
                            childPath += 2;
                            max_y = 0.5 * (min_y + max_y);
                        }
                        else {
                            min_y = 0.5 * (min_y + max_y);
                        }
#if DIM == 3
                        if (particles->z[tree->toDeleteNode[0] + bodyIndex + offset] < 0.5 * (min_z + max_z)) { // z direction
                            childPath += 4;
                            max_z = 0.5 * (min_z + max_z);
                        }
                        else {
                            min_z = 0.5 * (min_z + max_z);
                        }
#endif
#endif
                        atomicAdd(&tree->count[temp], 1); // ? do not count, since particles are just temporarily saved on this process
                        childIndex = tree->child[POW_DIM*temp + childPath];

                    }
#if DIM == 3
                    if (childIndex != -1) {
                        printf("[rank %i] (%f, %f, %f) vs (%f, %f, %f)\n", subDomainKeyTree->rank,
                               particles->x[tree->toDeleteNode[0] + bodyIndex + offset],
                               particles->y[tree->toDeleteNode[0] + bodyIndex + offset],
                               particles->z[tree->toDeleteNode[0] + bodyIndex + offset],
                               particles->x[childIndex],
                               particles->y[childIndex],
                               particles->z[childIndex]);
                        cudaAssert("insertReceivedPseudoParticles(): childIndex = %i temp = %i\n", childIndex, temp);
                    }
#endif

                    //insertionLevel++;

                    //temp = childIndex;
                    tree->child[POW_DIM*temp + childPath] = tree->toDeleteNode[0] + bodyIndex + offset;
                    //printf("[rank %i] gravity inserting POWDIM * %i + %i = %i (level = %i)\n", subDomainKeyTree->rank,
                    //       temp, childPath, tree->toDeleteNode[0] + bodyIndex + offset, level);

                    if (levels[bodyIndex + offset] != insertionLevel) {
                        // debug
                        //printf("[rank %i] index = %i childIndex = %i level = %i insertionLevel = %i path = %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i\n",
                        //       subDomainKeyTree->rank, tree->toDeleteNode[0] + bodyIndex + offset, childIndex,
                        //       levels[bodyIndex + offset], insertionLevel, path[0], path[1], path[2], path[3], path[4],
                        //       path[5], path[6], path[7], path[8], path[9], path[10]);
                        //printf("[rank %i] level = %i, insertionLevel = %i x = (%f, %f, %f) min/max = (%f, %f | %f, %f | %f, %f))\n", subDomainKeyTree->rank,
                        //       levels[bodyIndex + offset], insertionLevel,
                        //       particles->x[tree->toDeleteNode[0] + bodyIndex + offset],
                        //       particles->y[tree->toDeleteNode[0] + bodyIndex + offset],
                        //       particles->z[tree->toDeleteNode[0] + bodyIndex + offset],
                        //       min_x, max_x, min_y, max_y, min_z, max_z);
                        //printf("[rank %i] level = %i, insertionLevel = %i x = (%f, %f, %f) min/max = (%f, %f, %f))\n", subDomainKeyTree->rank,
                        //       levels[bodyIndex + offset], insertionLevel,
                        //       particles->x[tree->toDeleteNode[0] + bodyIndex + offset],
                        //       particles->y[tree->toDeleteNode[0] + bodyIndex + offset],
                        //       particles->z[tree->toDeleteNode[0] + bodyIndex + offset],
                        //       0.5 * (min_x + max_x), 0.5 * (min_y + max_y), 0.5 * (min_z + max_z));
                        //for (int i=0; i < (tree->toDeleteNode[1] - tree->toDeleteNode[0]); i++) {
                        //    printf("[rank %i] index = %i level = %i x = (%f, %f, %f) m = %f\n",
                        //            subDomainKeyTree->rank,
                        //            tree->toDeleteNode[0] + i,
                        //            levels[i],
                        //            particles->x[tree->toDeleteNode[0] + i],
                        //            particles->y[tree->toDeleteNode[0] + i],
                        //            particles->z[tree->toDeleteNode[0] + i],
                        //            particles->mass[tree->toDeleteNode[0] + i]);
                        //}

                        printf("insertReceivedPseudoParticles() for %i: level[%i] = %i != insertionLevel = %i!\n",
                               tree->toDeleteNode[0] + bodyIndex + offset, bodyIndex + offset,
                               levels[bodyIndex + offset], insertionLevel);
                        assert(0);
                    }
                }
                __threadfence();
                offset += stride;
            }
        }

        __global__ void insertReceivedParticles(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                                DomainList *domainList, DomainList *lowestDomainList, int n, int m) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;

            integer offset;

            real min_x, max_x;
#if DIM > 1
            real min_y, max_y;
#if DIM == 3
            real min_z, max_z;
#endif
#endif

            integer childPath;
            integer temp;

            offset = 0;

            bodyIndex += tree->toDeleteLeaf[0];

            while ((bodyIndex + offset) < tree->toDeleteLeaf[1]) { // && (bodyIndex + offset) >= tree->toDeleteLeaf[0]) {

                min_x = *tree->minX;
                max_x = *tree->maxX;
#if DIM > 1
                min_y = *tree->minY;
                max_y = *tree->maxY;
#if DIM == 3
                min_z = *tree->minZ;
                max_z = *tree->maxZ;
#endif
#endif

                temp = 0;
                childPath = 0;

                // find insertion point for body
                if (particles->x[bodyIndex + offset] < 0.5 * (min_x + max_x)) { // x direction
                    childPath += 1;
                    max_x = 0.5 * (min_x + max_x);
                }
                else {
                    min_x = 0.5 * (min_x + max_x);
                }
#if DIM > 1
                if (particles->y[bodyIndex + offset] < 0.5 * (min_y + max_y)) { // y direction
                    childPath += 2;
                    max_y = 0.5 * (min_y + max_y);
                }
                else {
                    min_y = 0.5 * (min_y + max_y);
                }
#if DIM == 3
                if (particles->z[bodyIndex + offset] < 0.5 * (min_z + max_z)) {  // z direction
                    childPath += 4;
                    max_z = 0.5 * (min_z + max_z);
                }
                else {
                    min_z = 0.5 * (min_z + max_z);
                }
#endif
#endif

                int childIndex = tree->child[temp*POW_DIM + childPath];

                // traverse tree until hitting leaf node
                while (childIndex >= m) {

                    temp = childIndex;

                    childPath = 0;

                    // find insertion point for body
                    if (particles->x[bodyIndex + offset] < 0.5 * (min_x + max_x)) { // x direction
                        childPath += 1;
                        max_x = 0.5 * (min_x + max_x);
                    }
                    else {
                        min_x = 0.5 * (min_x + max_x);
                    }
#if DIM > 1
                    if (particles->y[bodyIndex + offset] < 0.5 * (min_y + max_y)) { // y direction
                        childPath += 2;
                        max_y = 0.5 * (min_y + max_y);
                    }
                    else {
                        min_y = 0.5 * (min_y + max_y);
                    }
#if DIM == 3
                    if (particles->z[bodyIndex + offset] < 0.5 * (min_z + max_z)) { // z direction
                        childPath += 4;
                        max_z = 0.5 * (min_z + max_z);
                    }
                    else {
                        min_z = 0.5 * (min_z + max_z);
                    }
#endif
#endif
                    atomicAdd(&tree->count[temp], 1); // do not count, since particles are just temporarily saved on this process
                    childIndex = tree->child[POW_DIM*temp + childPath];

                }

                if (childIndex != -1) {
                    cudaAssert("ATTENTION: insertReceivedParticles(): childIndex = %i (%i, %i) (%i, %i)\n", childIndex,
                               tree->toDeleteLeaf[0], tree->toDeleteLeaf[1], tree->toDeleteNode[0],
                               tree->toDeleteNode[1]);
                    //printf("[rank %i] ATTENTION: childIndex = %i,... child[8 * %i + %i] = %i (%f, %f, %f) vs (%f, %f, %f)\n", subDomainKeyTree->rank,
                    //           childIndex, temp, childPath, bodyIndex + offset,
                    //           particles->x[childIndex], particles->y[childIndex], particles->z[childIndex],
                    //           particles->x[bodyIndex + offset], particles->y[bodyIndex + offset], particles->z[bodyIndex + offset]);

                }

                tree->child[POW_DIM*temp + childPath] = bodyIndex + offset;

                __threadfence();
                offset += stride;
            }

        }

        //TODO: do not delete children of lowest domain list nodes belonging to another process
        // but instead: delete the children of (lowest) domain list nodes if
        //  - toDeleteLeaf[0] < child < numParticles
        //  - child > toDeleteNode[0]
        // since problem for predictor-corrector decoupled gravity (having particles belonging to another process)
        __global__ void repairTree(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                   DomainList *domainList, DomainList *lowestDomainList,
                                   int n, int m, Curve::Type curveType) {

            integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
            integer stride = blockDim.x * gridDim.x;
            integer offset = 0;

            keyType key;
            int domainIndex;
            int proc;

            if (bodyIndex + offset == 0) {
                *tree->index = tree->toDeleteNode[0];
            }

            /*while ((bodyIndex + offset) < *lowestDomainList->domainListIndex) {
                domainIndex = lowestDomainList->domainListIndices[bodyIndex + offset];

                //key = tree->getParticleKey(particles, domainIndex, MAX_LEVEL, curveType); // working version
                //proc = subDomainKeyTree->key2proc(key);
                // //printf("[rank %i] deleting: proc = %i\n", subDomainKeyTree->rank, proc);
                //if (proc != subDomainKeyTree->rank) {
                //    for (int i=0; i<POW_DIM; i++) {
                //        //printf("[rank %i] deleting: POWDIM * %i + %i = %i\n", subDomainKeyTree->rank, domainIndex, i, tree->child[POW_DIM * domainIndex + i]);
                //        tree->child[POW_DIM * domainIndex + i] = -1;
                //    }
                //}

                for (int i=0; i<POW_DIM; i++) {
                    if (tree->child[POW_DIM * domainIndex + i] >= tree->toDeleteNode[0]) {
                        tree->child[POW_DIM * domainIndex + i] = -1;
                    }
                }
                offset += stride;
            }*/

            while ((bodyIndex + offset) < *domainList->domainListIndex) {
                domainIndex = domainList->domainListIndices[bodyIndex + offset];
                for (int i=0; i<POW_DIM; i++) {
                    if ((tree->child[POW_DIM * domainIndex + i] >= tree->toDeleteNode[0]
                    || (tree->child[POW_DIM * domainIndex + i] >= tree->toDeleteLeaf[0] &&
                    tree->child[POW_DIM * domainIndex + i] < n))
                    && particles->nodeType[tree->child[POW_DIM * domainIndex + i]] < 1) {

                        tree->child[POW_DIM * domainIndex + i] = -1;

                    }
                }
                offset += stride;
            }

            offset = tree->toDeleteLeaf[0];
            //delete inserted leaves
            while ((bodyIndex + offset) >= tree->toDeleteLeaf[0] && (bodyIndex + offset) < tree->toDeleteLeaf[1]) {
                for (int i=0; i<POW_DIM; i++) {
                    tree->child[(bodyIndex + offset)*POW_DIM + i] = -1;
                }
                tree->count[bodyIndex + offset] = 1;

                particles->x[bodyIndex + offset] = 0.;
                particles->vx[bodyIndex + offset] = 0.;
                particles->ax[bodyIndex + offset] = 0.;
                particles->g_ax[bodyIndex + offset] = 0.;
#if DIM > 1
                particles->y[bodyIndex + offset] = 0.;
                particles->vy[bodyIndex + offset] = 0.;
                particles->ay[bodyIndex + offset] = 0.;
                particles->g_ay[bodyIndex + offset] = 0.;
#if DIM == 3
                particles->z[bodyIndex + offset] = 0.;
                particles->vz[bodyIndex + offset] = 0.;
                particles->az[bodyIndex + offset] = 0.;
                particles->g_az[bodyIndex + offset] = 0.;
#endif
#endif
                particles->mass[bodyIndex + offset] = 0.;
                tree->start[bodyIndex + offset] = -1;
                tree->sorted[bodyIndex + offset] = 0;

                offset += stride;
            }

            offset = tree->toDeleteNode[0]; //0;
            //delete inserted cells
            while ((bodyIndex + offset) >= tree->toDeleteNode[0] && (bodyIndex + offset) < tree->toDeleteNode[1]) {
                for (int i=0; i<POW_DIM; i++) {
                    tree->child[(bodyIndex + offset)*POW_DIM + i] = -1;
                }
                tree->count[bodyIndex + offset] = 0;
                particles->x[bodyIndex + offset] = 0.;
                particles->vx[bodyIndex + offset] = 0.;
                particles->ax[bodyIndex + offset] = 0.;
#if DIM > 1
                particles->y[bodyIndex + offset] = 0.;
                particles->vy[bodyIndex + offset] = 0.;
                particles->ay[bodyIndex + offset] = 0.;
#if DIM == 3
                particles->z[bodyIndex + offset] = 0.;
                particles->vz[bodyIndex + offset] = 0.;
                particles->az[bodyIndex + offset] = 0.;
#endif
#endif
                particles->mass[bodyIndex + offset] = 0.;
                tree->start[bodyIndex + offset] = -1;
                tree->sorted[bodyIndex + offset] = 0;

                offset += stride;
            }
        }

        real Launch::globalCOM(Tree *tree, Particles *particles, real com[DIM]) {
            ExecutionPolicy executionPolicy(1, 1);
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::globalCOM, tree, particles, com);
        }

        real Launch::collectSendIndices(Tree *tree, Particles *particles, integer *sendIndices,
                                integer *particles2Send, integer *pseudoParticles2Send,
                                integer *pseudoParticlesLevel,
                                integer *particlesCount, integer *pseudoParticlesCount,
                                integer n, integer length, Curve::Type curveType) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::collectSendIndices, tree, particles, sendIndices,
                                particles2Send, pseudoParticles2Send, pseudoParticlesLevel, particlesCount,
                                pseudoParticlesCount, n, length, curveType);
        }

        real Launch::testSendIndices(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                             integer *sendIndices, integer *markedSendIndices,
                             integer *levels, Curve::Type curveType, integer length) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::testSendIndices, subDomainKeyTree,
                                tree, particles, sendIndices, markedSendIndices, levels, curveType, length);
        }

        real Launch::zeroDomainListNodes(Particles *particles, DomainList *domainList, DomainList *lowestDomainList) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::zeroDomainListNodes, particles, domainList,
                                lowestDomainList);
        }

        real Launch::prepareLowestDomainExchange(Particles *particles, DomainList *lowestDomainList,
                                                 Helper *helper, Entry::Name entry) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::prepareLowestDomainExchange, particles,
                                lowestDomainList, helper, entry);
        }

        real Launch::updateLowestDomainListNodes(Particles *particles, DomainList *lowestDomainList,
                                                 Helper *helper, Entry::Name entry) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::updateLowestDomainListNodes, particles,
                                lowestDomainList, helper, entry);

        }

        real Launch::compLowestDomainListNodes(Tree *tree, Particles *particles, DomainList *lowestDomainList) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::compLowestDomainListNodes, tree, particles,
                                lowestDomainList);
        }

        real Launch::compLocalPseudoParticles(Tree *tree, Particles *particles, DomainList *domainList, int n) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::compLocalPseudoParticles, tree, particles,
                                domainList, n);
        }

        real Launch::compDomainListPseudoParticlesPerLevel(Tree *tree, Particles *particles, DomainList *domainList,
                                                   DomainList *lowestDomainList, int n, int level) {
            ExecutionPolicy executionPolicy(256, 1);
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::compDomainListPseudoParticlesPerLevel, tree,
                                particles, domainList, lowestDomainList, n, level);
        }

        real Launch::compDomainListPseudoParticles(Tree *tree, Particles *particles, DomainList *domainList,
                                                   DomainList *lowestDomainList, int n) {
            ExecutionPolicy executionPolicy(256, 1);
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::compDomainListPseudoParticles, tree,
                                particles, domainList, lowestDomainList, n);
        }

        real Launch::computeForces_v1(Tree *tree, Particles *particles, real radius, integer n, integer m,
                                      SubDomainKeyTree *subDomainKeyTree, real theta, real smoothing) {
            size_t sharedMemory = sizeof(real) * MAX_DEPTH;
            ExecutionPolicy executionPolicy(256, 256, sharedMemory);
            //ExecutionPolicy executionPolicy(512, 256, sharedMemory);
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::computeForces_v1, tree, particles,
                                radius, n, m, subDomainKeyTree, theta, smoothing);
        }

        real Launch::computeForces_v1_1(Tree *tree, Particles *particles, real radius, integer n, integer m,
                                        SubDomainKeyTree *subDomainKeyTree, real theta, real smoothing) {
            size_t sharedMemory = sizeof(real) * MAX_DEPTH;
            ExecutionPolicy executionPolicy(256, 256, sharedMemory);
            //ExecutionPolicy executionPolicy(512, 256, sharedMemory);
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::computeForces_v1_1, tree, particles,
                                radius, n, m, subDomainKeyTree, theta, smoothing);
        }

        real Launch::computeForces_v1_2(Tree *tree, Particles *particles, real radius, integer n, integer m,
                                      SubDomainKeyTree *subDomainKeyTree, real theta, real smoothing) {
            size_t sharedMemory = (2*sizeof(int) + sizeof(real)) * MAX_DEPTH;
            ExecutionPolicy executionPolicy(256, 256, sharedMemory);
            //ExecutionPolicy executionPolicy(512, 256, sharedMemory);
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::computeForces_v1_2, tree, particles,
                                radius, n, m, subDomainKeyTree, theta, smoothing);
        }

        real Launch::computeForces_v2(Tree *tree, Particles *particles, real radius, integer n, integer m,
                                      integer blockSize, integer warp, integer stackSize,
                                      SubDomainKeyTree *subDomainKeyTree, real theta,
                                      real smoothing) {

            size_t sharedMemory = (sizeof(real)+sizeof(integer))*stackSize*blockSize/warp;
            //size_t sharedMemory = 2*sizeof(real)*stackSize*blockSize/warp;
            ExecutionPolicy executionPolicy(256, 256, sharedMemory);
            //ExecutionPolicy executionPolicy(512, 256, sharedMemory);
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::computeForces_v2, tree, particles, radius,
                                n, m, blockSize, warp, stackSize, subDomainKeyTree, theta, smoothing);
        }

        real Launch::computeForces_v2_1(Tree *tree, Particles *particles, integer n, integer m, integer blockSize,
                                        integer warp, integer stackSize, SubDomainKeyTree *subDomainKeyTree,
                                        real theta, real smoothing) {

            size_t sharedMemory = (sizeof(real)+sizeof(integer))*stackSize*blockSize/warp;
            //size_t sharedMemory = 2*sizeof(real)*stackSize*blockSize/warp;
            ExecutionPolicy executionPolicy(256, 256, sharedMemory);
            //ExecutionPolicy executionPolicy(512, 256, sharedMemory);
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::computeForces_v2_1, tree, particles, n, m,
                                blockSize, warp, stackSize, subDomainKeyTree, theta, smoothing);
        }

        //real Launch::symbolicForce(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
        //                   DomainList *domainList, integer *sendIndices,
        //                   real diam, real theta_, integer n, integer m, integer relevantIndex,
        //                   Curve::Type curveType) {
        //    ExecutionPolicy executionPolicy(1, 256);
        //    return cuda::launch(true, executionPolicy, ::Gravity::Kernel::symbolicForce, subDomainKeyTree, tree,
        //                        particles, domainList, sendIndices, diam, theta_, n, m, relevantIndex, curveType);
        //}

        real Launch::intermediateSymbolicForce(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                   DomainList *domainList, integer *sendIndices, real diam, real theta_,
                                   integer n, integer m, integer relevantIndex, integer level,
                                   Curve::Type curveType) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::intermediateSymbolicForce, subDomainKeyTree, tree,
                                particles, domainList, sendIndices, diam, theta_, n, m, relevantIndex, level, curveType);
        }

        real Launch::symbolicForce(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                           DomainList *domainList, integer *sendIndices, real diam, real theta_,
                           integer n, integer m, integer relevantIndex, integer level,
                           Curve::Type curveType) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::symbolicForce, subDomainKeyTree, tree,
                                particles, domainList, sendIndices, diam, theta_, n, m, relevantIndex, level, curveType);
        }

        real Launch::compTheta(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                       DomainList *domainList, Helper *helper, Curve::Type curveType) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::compTheta, subDomainKeyTree, tree, particles,
                                domainList, helper, curveType);
        }

        real Launch::createKeyHistRanges(Helper *helper, integer bins) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::createKeyHistRanges, helper, bins);
        }

        real Launch::keyHistCounter(Tree *tree, Particles *particles, SubDomainKeyTree *subDomainKeyTree,
                            Helper *helper, int bins, int n, Curve::Type curveType) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::keyHistCounter, tree, particles,
                                subDomainKeyTree, helper, bins, n, curveType);
        }

        real Launch::calculateNewRange(SubDomainKeyTree *subDomainKeyTree, Helper *helper, int bins, int n,
                               Curve::Type curveType) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::calculateNewRange, subDomainKeyTree, helper,
                                bins, n, curveType);
        }

        //real Launch::insertReceivedPseudoParticles(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
        //                                   integer *levels, int n, int m) {
        //    ExecutionPolicy executionPolicy(1, 256);
        //    return cuda::launch(true, executionPolicy, ::Gravity::Kernel::insertReceivedPseudoParticles,
        //                        subDomainKeyTree, tree, particles, levels, n, m);
        //}

        real Launch::insertReceivedPseudoParticles(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                                   integer *levels, int level, int n, int m) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::insertReceivedPseudoParticles,
                                subDomainKeyTree, tree, particles, levels, level, n, m);
        }

        real Launch::insertReceivedParticles(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                     DomainList *domainList, DomainList *lowestDomainList, int n, int m) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::insertReceivedParticles, subDomainKeyTree,
                                tree, particles, domainList, lowestDomainList, n, m);
        }

        real Launch::repairTree(SubDomainKeyTree *subDomainKeyTree, Tree *tree, Particles *particles,
                                DomainList *domainList, DomainList *lowestDomainList,
                                int n, int m, Curve::Type curveType) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::Gravity::Kernel::repairTree, subDomainKeyTree, tree,
                                particles, domainList, lowestDomainList, n, m, curveType);
        }

    }
}
