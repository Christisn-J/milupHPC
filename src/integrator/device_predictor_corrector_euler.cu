#include "hip/hip_runtime.h"
#include "../../include/integrator/device_predictor_corrector_euler.cuh"
#include "../../include/cuda_utils/cuda_launcher.cuh"

namespace PredictorCorrectorEulerNS {


    CUDA_CALLABLE_MEMBER Shared::Shared() {

    }
    CUDA_CALLABLE_MEMBER Shared::Shared(real *forces, real *courant, real *artVisc) {
        this->forces = forces;
        this->courant = courant;
        this->artVisc = artVisc;
    }
    CUDA_CALLABLE_MEMBER Shared::~Shared() {

    }
    CUDA_CALLABLE_MEMBER void Shared::set(real *forces, real *courant, real *artVisc) {
        this->forces = forces;
        this->courant = courant;
        this->artVisc= artVisc;
    }
    CUDA_CALLABLE_MEMBER void Shared::setE(real *e) {
        this->e = e;
    }
    CUDA_CALLABLE_MEMBER void Shared::setRho(real *rho) {
        this->rho = rho;
    }
    CUDA_CALLABLE_MEMBER void Shared::setVmax(real *vmax) {
        this->vmax = vmax;
    }
    namespace SharedNS {
        __global__ void set(Shared *shared, real *forces, real *courant, real *artVisc) {
            shared->set(forces, courant, artVisc);
        }
        __global__ void setE(Shared *shared, real *e) {
            shared->setE(e);
        }
        __global__ void setRho(Shared *shared, real *rho) {
            shared->setRho(rho);
        }
        __global__ void setVmax(Shared *shared, real *vmax) {
            shared->setVmax(vmax);
        }
        namespace Launch {
            void set(Shared *shared, real *forces, real *courant, real *artVisc) {
                ExecutionPolicy executionPolicy(1, 1);
                cuda::launch(false, executionPolicy, ::PredictorCorrectorEulerNS::SharedNS::set, shared,
                             forces, courant, artVisc);
            }
            void setE(Shared *shared, real *e) {
                ExecutionPolicy executionPolicy(1, 1);
                cuda::launch(false, executionPolicy, ::PredictorCorrectorEulerNS::SharedNS::setE, shared, e);
            }
            void setRho(Shared *shared, real *rho) {
                ExecutionPolicy executionPolicy(1, 1);
                cuda::launch(false, executionPolicy, ::PredictorCorrectorEulerNS::SharedNS::setRho, shared, rho);
            }
            void setVmax(Shared *shared, real *vmax) {
                ExecutionPolicy executionPolicy(1, 1);
                cuda::launch(false, executionPolicy, ::PredictorCorrectorEulerNS::SharedNS::setVmax, shared, vmax);
            }
        }
    }

    CUDA_CALLABLE_MEMBER BlockShared::BlockShared() {

    }
    CUDA_CALLABLE_MEMBER BlockShared::BlockShared(real *forces, real *courant, real *artVisc) {
        this->forces = forces;
        this->courant = courant;
        this->artVisc = artVisc;
    }
    CUDA_CALLABLE_MEMBER BlockShared::~BlockShared() {

    }
    CUDA_CALLABLE_MEMBER void BlockShared::set(real *forces, real *courant, real *artVisc) {
        this->forces = forces;
        this->courant = courant;
        this->artVisc= artVisc;
    }
    CUDA_CALLABLE_MEMBER void BlockShared::setE(real *e) {
        this->e = e;
    }
    CUDA_CALLABLE_MEMBER void BlockShared::setRho(real *rho) {
        this->rho = rho;
    }
    CUDA_CALLABLE_MEMBER void BlockShared::setVmax(real *vmax) {
        this->vmax = vmax;
    }
    namespace BlockSharedNS {
        __global__ void set(BlockShared *blockShared, real *forces, real *courant, real *artVisc) {
            blockShared->set(forces, courant, artVisc);
        }
        __global__ void setE(BlockShared *blockShared, real *e) {
            blockShared->setE(e);
        }
        __global__ void setRho(BlockShared *blockShared, real *rho) {
            blockShared->setRho(rho);
        }
        __global__ void setVmax(BlockShared *blockShared, real *vmax) {
            blockShared->setVmax(vmax);
        }

        namespace Launch {
            void set(BlockShared *blockShared, real *forces, real *courant, real *artVisc) {
                ExecutionPolicy executionPolicy(1, 1);
                cuda::launch(false, executionPolicy, ::PredictorCorrectorEulerNS::BlockSharedNS::set, blockShared,
                             forces, courant, artVisc);
            }
            void setE(BlockShared *blockShared, real *e) {
                ExecutionPolicy executionPolicy(1, 1);
                cuda::launch(false, executionPolicy, ::PredictorCorrectorEulerNS::BlockSharedNS::setE, blockShared, e);
            }
            void setRho(BlockShared *blockShared, real *rho) {
                ExecutionPolicy executionPolicy(1, 1);
                cuda::launch(false, executionPolicy, ::PredictorCorrectorEulerNS::BlockSharedNS::setRho,
                             blockShared, rho);
            }
            void setVmax(BlockShared *blockShared, real *vmax) {
                ExecutionPolicy executionPolicy(1, 1);
                cuda::launch(false, executionPolicy, ::PredictorCorrectorEulerNS::BlockSharedNS::setVmax,
                             blockShared, vmax);
            }
        }
    }

    namespace Kernel {

        __global__ void corrector(Particles *particles, IntegratedParticles *predictor, real dt, int numParticles) {

            int i;
            // particle loop
            for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {

// just for debugging purposes!!!
/*
                particles->vx[i] += dt * (particles->ax[i] + particles->g_ax[i]);
#if DIM > 1
                particles->vy[i] += dt * (particles->ay[i] + particles->g_ay[i]);
#if DIM == 3
                particles->vz[i] += dt * (particles->az[i] + particles->g_az[i]);
#endif
#endif

                // calculating/updating the positions
                particles->x[i] += dt * particles->vx[i];
#if DIM > 1
                particles->y[i] += dt * particles->vy[i];
#if DIM == 3
                particles->z[i] += dt * particles->vz[i];
#endif
#endif
*/
// end: just for debugging purposes!!!

                particles->x[i] = particles->x[i] + dt/2 * (predictor->vx[i] + particles->vx[i]);
                //if (i == 12) { //(i % 1000 == 0) {
                //    printf("corrector: x[%i] = %e + %e/2 * (%e + %e)\n", i, particles->x[i], dt, predictor->vx[i],
                //           particles->vx[i]);
                //}
                particles->vx[i] = particles->vx[i] + dt/2 * (predictor->ax[i] + particles->ax[i] + 2 * particles->g_ax[i]);
                //if (i == 12) { //(i % 1000 == 0) {
                //    printf("corrector: vx[%i] = %e + %e/2 * (%e + %e + 2 * %e)\n", i, particles->vx[i], dt, predictor->ax[i],
                //           particles->ax[i], particles->g_ax);
                //}
                particles->ax[i] = 0.5 * (predictor->ax[i] + particles->ax[i]) + particles->g_ax[i];
                //if (i == 12) { //(i % 1000 == 0) {
                //    printf("corrector: ax[%i] = 1/2 * (%e + %e) + %e)\n", i, predictor->ax[i], particles->ax[i], particles->g_ax);
                //}
#if DIM > 1
                particles->y[i] = particles->y[i] + dt/2 * (predictor->vy[i] + particles->vy[i]);
                particles->vy[i] = particles->vy[i] + dt/2 * (predictor->ay[i] + particles->ay[i] + 2 * particles->g_ay[i]);
                particles->ay[i] = 0.5 * (predictor->ay[i] + particles->ay[i]) + particles->g_ay[i];
#if DIM == 3
                particles->z[i] = particles->z[i] + dt/2 * (predictor->vz[i] + particles->vz[i]);
                particles->vz[i] = particles->vz[i] + dt/2 * (predictor->az[i] + particles->az[i] + 2 * particles->g_az[i]);
                particles->az[i] = 0.5 * (predictor->az[i] + particles->az[i]) + particles->g_az[i];
#endif
#endif

// TODO: some SPH flag?
#if INTEGRATE_DENSITY
                particles->rho[i] = particles->rho[i] + dt/2 * (predictor->drhodt[i] + particles->drhodt[i]);
                particles->drhodt[i] = 0.5 * (predictor->drhodt[i] + particles->drhodt[i]); // not needed? Debugging purposes?
                //if (i == 12) { //(i % 1000 == 0) {
                //    printf("corrector: rho[%i] = %e + %e/2 * (%e + %e)\n", i, particles->rho[i], dt, predictor->drhodt[i],
                //           particles->drhodt[i]);
                //}
#else
                //p.rho[i] = p.rho[i];
#endif
#if INTEGRATE_ENERGY
                particles->e[i] = particles->e[i] + dt/2 * (predictor->dedt[i] + particles->dedt[i]);
                if (particles->e[i] < 1e-6) {
                    particles->e[i] = 1e-6;
                }
                particles->dedt[i] = 0.5 * (predictor->dedt[i] + particles->dedt[i]);
                //if (i == 12) { //(i % 1000 == 0) {
                //    printf("corrector: e[%i] = %e + %e/2 * (%e + %e)\n", i, particles->e[i], dt, predictor->dedt[i],
                //           particles->dedt[i]);
                //}
#endif
#if INTEGRATE_SML
#if DECOUPLE_SML
                particles->sml[i] = particles->sml[i] + dt * particles->dsmldt[i];
                //particles->dsmldt[i] = particles->dsmldt[i];
#else
                particles->sml[i] = particles->sml[i] + dt/2 * (predictor->dsmldt[i] + particles->dsmldt[i]);
                particles->dsmldt[i] = 0.5 * (predictor->dsmldt[i] + particles->dsmldt[i]);
#endif
#else
                particles->sml[i] = predictor->sml[i];
#endif
                //if (i % 1000 == 0) {
                //    printf("i: %i, particles->cs = %e, predictor->cs = %e\n", i, particles->cs[i], predictor->cs[i]);
                //}
                // TODO: needed?
                //predictor->reset(i); //TODO: move somewhere else?
            }
        }

        __global__ void predictor(Particles *particles, IntegratedParticles *predictor, real dt, int numParticles) {

            int i;

            for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {

                predictor->x[i] = particles->x[i] + dt * particles->vx[i];
                predictor->vx[i] = particles->vx[i] + dt * (particles->ax[i] + particles->g_ax[i]);
#if DIM > 1
                predictor->y[i] = particles->y[i] + dt * particles->vy[i];
                predictor->vy[i] = particles->vy[i] + dt * (particles->ay[i] + particles->g_ay[i]);
#if DIM == 3
                predictor->z[i] = particles->z[i] + dt * particles->vz[i];
                predictor->vz[i] = particles->vz[i] + dt * (particles->az[i] + particles->g_az[i]);
#endif
#endif

// TODO: some SPH flag?
#if INTEGRATE_DENSITY
                predictor->rho[i] = particles->rho[i] + dt * particles->drhodt[i];
                //predictor->drhodt[i] = particles->drhodt[i];
#else
                //predictor->rho[i] = particles->rho[i];
#endif
//TODO: add other quantities --> solids
#if INTEGRATE_ENERGY
                predictor->e[i] = particles->e[i] + dt * particles->dedt[i];
                // TODO: in principle there should not be a energy floor (but needed for sedov)
                if (predictor->e[i] < 1e-6) {
                    predictor->e[i] = 1e-6;
                }
#endif
#if INTEGRATE_SML
#if DECOUPLE_SML
                predictor->sml[i] = particles->sml[i] + dt * particles->dsmldt[i];
#else
                predictor->sml[i] = particles->sml[i];
#endif
#else
                predictor->sml[i] = particles->sml[i];
#endif
                predictor->cs[i] = particles->cs[i];
                // TODO: why is this needed?
                predictor->p[i] = particles->p[i];
                //predictor->ax[i] = particles->ax[i];
                //predictor->ay[i] = particles->ay[i];
                //predictor->az[i] = particles->az[i];
            }

        }

        /**
         * Conditions to be applied:
         *
         * * sound waves traveling faster than a fraction of the smoothing length
         *     * $\Delta t \leq C \frac{h}{c + 1.2 (\alpha_{\nu} c + \beta_{\nu} \mu_{max})}$
         *     * where $c$ is the sound speed; $\alpha_{\nu}$ and $\beta_{\nu}$ are the viscosity parameters, $\mu_{max}$ is the maximal value of $\mu_{ij}$ and $C$ is the Courant number
         * * time step constrains the distance a particle travels due to acceleration
         *     * $\Delta t \leq \sqrt{\frac{h}{|\vec{a}|}}$
         * * all other quantities $f$ have to be prevented from growing too fast within one time step
         *     * $\Delta t \leq \begin{cases} a \frac{|f| + f_{min}}{|df|} & |df| > 0 \\ \Delta t_{max} & |df| = 0 \\ \end{cases}$ where $a < 1$
         * * additional constraint regarding parallelization approach:
         *     * particles should not move further than h/2
         *     * $\Delta t \cdot v_{max} < \frac{h}{2} \, \Leftrightarrow \Delta t < \frac{h}{2 v_{max}}$
         */
        __global__ void setTimeStep(SimulationTime *simulationTime, Material *materials, Particles *particles,
                                    BlockShared *blockShared, int *blockCount, real searchRadius, int numParticles) {

#define SAFETY_FIRST 0.1

            __shared__ real sharedForces[NUM_THREADS_LIMIT_TIME_STEP];
            __shared__ real sharedCourant[NUM_THREADS_LIMIT_TIME_STEP];
            __shared__ real sharedArtVisc[NUM_THREADS_LIMIT_TIME_STEP];
            __shared__ real sharede[NUM_THREADS_LIMIT_TIME_STEP];
            __shared__ real sharedrho[NUM_THREADS_LIMIT_TIME_STEP];
            __shared__ real sharedVmax[NUM_THREADS_LIMIT_TIME_STEP];

            int i, j, k, m;
            int d, dd;
            int index;
#if INTEGRATE_ENERGY
            bool hasEnergy;
#endif
            real forces = DBL_MAX;
            real courant = DBL_MAX;
            real dtx = DBL_MAX;
            real dtrho = DBL_MAX;
            real dte = DBL_MAX;
            real vmax = 0.; //TODO: initial value
            real temp;
            real sml;
            int matId;

            real ax;
#if DIM > 1
            real ay;
#if DIM == 3
            real az;
#endif
#endif
            real dtartvisc = DBL_MAX;

            for (i = threadIdx.x + blockIdx.x * blockDim.x; i < numParticles; i+= blockDim.x * gridDim.x) {

                matId = particles->materialId[i];

#if INTEGRATE_ENERGY
            hasEnergy = false;

//          switch  (matEOS[matId]) {
//              case (EOS_TYPE_TILLOTSON):
//                  hasEnergy = true;
//                  break;
//              case (EOS_TYPE_JUTZI):
//                  hasEnergy = true;
//                  break;
//		  	    case (EOS_TYPE_JUTZI_ANEOS):
//		  		    hasEnergy = true;
//		  		    break;
//              case (EOS_TYPE_SIRONO):
//                  hasEnergy = true;
//                  break;
//              case (EOS_TYPE_EPSILON):
//                  hasEnergy = true;
//                  break;
//              case (EOS_TYPE_ANEOS):
//                  hasEnergy = true;
//                  break;
//              default:
//                  hasEnergy = false;
//                  break;
//          }
#endif
                ax = 0.;
#if DIM > 1
                ay = 0.;
#if DIM == 3
                az = 0.;
#endif
#endif
#if GRAVITY_SIM
                ax += particles->g_ax[i];
#if DIM > 1
                ay += particles->g_ay[i];
#if DIM == 3
                az += particles->g_az[i];
#endif
#endif
#endif
#if SPH_SIM
                ax += particles->ax[i];
#if DIM > 1
                ay += particles->ay[i];
#if DIM == 3
                az += particles->az[i];
#endif
#endif
#endif
                temp = ax * ax;
#if DIM > 1
                temp += ay * ay;
#if DIM == 3
                temp += az * az;
#endif
#endif

                //if (i % 10000 == 0) {
                //    printf("i: %i ax = %e, ay = %e, az = %e\n", i, ax, ay, az);
                //}
                sml = particles->sml[i];
                temp = cuda::math::sqrt(sml / cuda::math::sqrt(temp));
                forces = cuda::math::min(forces, temp);
                //if (forces == 0.) {
                //    printf("forces: %e, sml: %e, temp: %e ax = %e, g_ax = %e (noi: %i)\n", forces, sml, temp, particles->ax[i],
                //           particles->g_ax[i], particles->noi[i]);
                //}
                temp = sml / particles->cs[i];
                courant = cuda::math::min(courant, temp);

                temp = COURANT_FACT * sml / (particles->cs[i] + 1.2 * materials[matId].artificialViscosity.alpha * particles->cs[i] +
                            materials[matId].artificialViscosity.beta * particles->muijmax[i]);
                dtartvisc = min(dtartvisc, temp);

#if DIM == 1
                temp = cuda::math::sqrt(particles->vx[i] * particles->vx[i]);
#elif DIM == 2
                temp = cuda::math::sqrt(particles->vx[i] * particles->vx[i] +
                                        particles->vy[i] * particles->vy[i]);
#else
                temp = cuda::math::sqrt(particles->vx[i] * particles->vx[i] +
                                        particles->vy[i] * particles->vy[i] +
                                        particles->vz[i] * particles->vz[i]);
#endif
                //if (i % 10000 == 0) {
                //    printf("i: %i vx = %e, vy = %e, vz = %e\n", i, particles->vx[i], particles->vy[i], particles->vz[i]);
                //}

                vmax = cuda::math::max(temp, vmax);

#if INTEGRATE_DENSITY
                if (particles->drhodt[i] != 0) {
                    //TODO: define rhomin_d
                    double rhomin_d = 0.01;
                    temp = SAFETY_FIRST * (cuda::math::abs(particles->rho[i])+rhomin_d)/cuda::math::abs(particles->drhodt[i]);
                    dtrho = cuda::math::min(temp, dtrho);
                }
#endif
#if INTEGRATE_ENERGY
                //if (particles->dedt[i] != 0 && hasEnergy) {
                    //TODO: define emin_d
                    //temp = SAFETY_FIRST * (cuda::math::abs(particles->e[i])+emin_d)/cuda::math::abs(particles->dedt[i]);
                    //dte = cuda::math::min(temp, dte);
                //}
#endif

            }

            __threadfence();

            i = threadIdx.x;
            sharedForces[i] = forces;
            sharedCourant[i] = courant;
            sharede[i] = dte;
            sharedrho[i] = dtrho;
            sharedArtVisc[i] = dtartvisc;
            sharedVmax[i] = vmax;

            for (j = NUM_THREADS_LIMIT_TIME_STEP / 2; j > 0; j /= 2) {
                __syncthreads();
                if (i < j) {
                    k = i + j;
                    sharedForces[i] = forces = cuda::math::min(forces, sharedForces[k]);
                    sharedCourant[i] = courant = cuda::math::min(courant, sharedCourant[k]);
                    sharede[i] = dte = cuda::math::min(dte, sharede[k]);
                    sharedrho[i] = dtrho = cuda::math::min(dtrho, sharedrho[k]);
                    sharedArtVisc[i] = dtartvisc = cuda::math::min(dtartvisc, sharedArtVisc[k]);
                    sharedVmax[i] = vmax = cuda::math::max(vmax, sharedVmax[k]);
                }
            }
            // write block result to global memory
            if (i == 0) {
                k = blockIdx.x;
                blockShared->forces[k] = forces;
                blockShared->courant[k] = courant;
                blockShared->e[k] = dte;
                blockShared->rho[k] = dtrho;
                blockShared->artVisc[k] = dtartvisc;
                blockShared->vmax[k] = vmax;


                m = gridDim.x - 1;
                if (m == atomicInc((unsigned int *)blockCount, m)) {
                    // last block, so combine all block results
                    for (j = 0; j <= m; j++) {
                        forces = cuda::math::min(forces, blockShared->forces[j]);
                        courant = cuda::math::min(courant, blockShared->courant[j]);
                        dte = cuda::math::min(dte, blockShared->e[j]);
                        dtrho = cuda::math::min(dtrho, blockShared->rho[j]);
                        dtartvisc = cuda::math::min(dtartvisc, blockShared->artVisc[j]);
                        vmax = cuda::math::min(vmax, blockShared->vmax[j]);
                    }
                    // set new timestep
                    *simulationTime->dt = dtx = cuda::math::min(COURANT_FACT*courant, FORCES_FACT*forces);
                    //printf("courant: dt = %e (courant = %e)\n", COURANT_FACT*courant, courant);
                    //printf("force  : dt = %e (forces = %e)\n", FORCES_FACT*forces, forces);

                    if (vmax > 0. && searchRadius > 0.) { // TODO: searchRadius = 0 for 1 process
                        *simulationTime->dt = cuda::math::min(*simulationTime->dt, searchRadius / (2 * vmax));
                        //printf("search : dt = %e (vmax = %e)\n", searchRadius / (2 * vmax), vmax);
                    }
#if INTEGRATE_ENERGY
                    *simulationTime->dt = cuda::math::min(*simulationTime->dt, dte);
                    //printf("dte: %e\n", dte);
#endif
#if INTEGRATE_DENSITY
                    *simulationTime->dt = cuda::math::min(*simulationTime->dt, dtrho);
                    //printf("dtrho: %e\n", dtrho);
#endif

                    *simulationTime->dt = cuda::math::min(*simulationTime->dt, dtartvisc);
                    //printf("viscos : dt = %e\n", dtartvisc);

                    *simulationTime->dt = cuda::math::min(*simulationTime->dt, *simulationTime->subEndTime - *simulationTime->currentTime);
                    if (*simulationTime->dt > *simulationTime->dt_max) {
                        *simulationTime->dt = *simulationTime->dt_max;
                    }
                    //if (*simulationTime->dt < 1.e8) {
                    //    *simulationTime->dt = 1.e8;
                    //}
                    //printf("max    : dt = %e\n", *simulationTime->dt_max);
                    //printf("dt: %e\n", *simulationTime->dt);

                    //printf("Time Step Information: dt(v and x): %.17e dtS: %.17e dte: %.17e dtrho: %.17e dtdamage: %.17e dtalpha: %.17e dtalpha_epspor: %.17e dtepsilon_v: %.17e\n", dtx, dtS, dte, dtrho, dtdamage, dtalpha, dtalpha_epspor, dtepsilon_v);
                    //printf("time: %.17e timestep set to %.17e, integrating until %.17e \n", currentTimeD, dt, endTimeD);

                    // reset block count
                    *blockCount = 0;
                }
            }
        }

        real Launch::corrector(Particles *particles, IntegratedParticles *predictor, real dt, int numParticles) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::PredictorCorrectorEulerNS::Kernel::corrector, particles,
                                predictor, dt, numParticles);
        }
        real Launch::predictor(Particles *particles, IntegratedParticles *predictor, real dt, int numParticles) {
            ExecutionPolicy executionPolicy;
            return cuda::launch(true, executionPolicy, ::PredictorCorrectorEulerNS::Kernel::predictor, particles,
                                predictor, dt, numParticles);
        }

        real Launch::setTimeStep(int multiProcessorCount, SimulationTime *simulationTime, Material *materials, Particles *particles,
                                 BlockShared *blockShared, int *blockCount, real searchRadius, int numParticles) {
            ExecutionPolicy executionPolicy(multiProcessorCount, 256);
            return cuda::launch(true, executionPolicy, ::PredictorCorrectorEulerNS::Kernel::setTimeStep, simulationTime,
                                materials, particles, blockShared, blockCount, searchRadius, numParticles);
        }

        real Launch::pressureChangeCheck() {

        }

    }
}
