#include "hip/hip_runtime.h"
#include "../../include/integrator/device_explicit_euler.cuh"
#include "../../include/cuda_utils/cuda_launcher.cuh"

__global__ void ExplicitEulerNS::Kernel::update(Particles *particles, integer n, real dt) {

    integer bodyIndex = threadIdx.x + blockIdx.x * blockDim.x;
    integer stride = blockDim.x * gridDim.x;
    integer offset = 0;

    while (bodyIndex + offset < n) {

        // calculating/updating the velocities
        particles->vx[bodyIndex + offset] += dt * (particles->ax[bodyIndex + offset] + particles->g_ax[bodyIndex + offset]);
        //if ((bodyIndex + offset) % 1000 == 0) {
        //    printf("vx[%i] += dt * (%f + %f) = %f\n", bodyIndex + offset, particles->ax[bodyIndex + offset],
        //           particles->g_ax[bodyIndex + offset], particles->vx[bodyIndex + offset]);
        //}
#if DIM > 1
        particles->vy[bodyIndex + offset] += dt * (particles->ay[bodyIndex + offset] + particles->g_ay[bodyIndex + offset]);
#if DIM == 3
        particles->vz[bodyIndex + offset] += dt * (particles->az[bodyIndex + offset] + particles->g_az[bodyIndex + offset]);
#endif
#endif

        // calculating/updating the positions
        particles->x[bodyIndex + offset] += dt * particles->vx[bodyIndex + offset];
#if DIM > 1
        particles->y[bodyIndex + offset] += dt * particles->vy[bodyIndex + offset];
#if DIM == 3
        particles->z[bodyIndex + offset] += dt * particles->vz[bodyIndex + offset];
#endif
#endif

        // debug
        //if (bodyIndex + offset == n - 1 || bodyIndex + offset == 0) {
        // //if ((bodyIndex + offset) % 100 == 0) {
        //    printf("update: %i (%f, %f, %f) x += (%f, %f, %f)\n", bodyIndex + offset, particles->x[bodyIndex + offset],
        //           particles->y[bodyIndex + offset], particles->z[bodyIndex + offset], d * dt * particles->vx[bodyIndex + offset],
        //           d * dt * particles->vy[bodyIndex + offset], d * dt * particles->vz[bodyIndex + offset]);
        //    printf("update: %i (%f, %f, %f) %f (%f, %f, %f) (%f, %f, %f) %f\n", bodyIndex + offset,
        //           particles->x[bodyIndex + offset],
        //           particles->y[bodyIndex + offset],
        //           particles->z[bodyIndex + offset],
        //           particles->mass[bodyIndex + offset],
        //           particles->vx[bodyIndex + offset],
        //           particles->vy[bodyIndex + offset],
        //           particles->vz[bodyIndex + offset],
        //           particles->ax[bodyIndex + offset],
        //           particles->ay[bodyIndex + offset],
        //           particles->az[bodyIndex + offset],
        //           particles->ax[bodyIndex + offset] * particles->ax[bodyIndex + offset] +
        //           particles->ay[bodyIndex + offset] * particles->ay[bodyIndex + offset] +
        //           particles->az[bodyIndex + offset] * particles->az[bodyIndex + offset]);
        //}
        //if (abs(particles->x[bodyIndex + offset]) < 3 && abs(particles->y[bodyIndex + offset]) < 3 &&
        //        abs(particles->z[bodyIndex + offset]) < 3) {
        //    printf("centered: index = %i (%f, %f, %f) %f\n", bodyIndex + offset,
        //           particles->x[bodyIndex + offset],
        //           particles->y[bodyIndex + offset],
        //           particles->z[bodyIndex + offset],
        //           particles->mass[bodyIndex + offset]);
        //    if (particles->mass[bodyIndex + offset] < 1) {
        //        //assert(0);
        //    }
        //}
        //if (abs(particles->ax[bodyIndex + offset]) < 10 && abs(particles->ay[bodyIndex + offset]) < 10 &&
        //    abs(particles->az[bodyIndex + offset]) < 10) {
        //if (true) {
        //    printf("ACCELERATION tiny! centered: index = %i (%f, %f, %f) %f (%f, %f, %f) (%f, %f, %f)\n", bodyIndex + offset,
        //           particles->x[bodyIndex + offset],
        //           particles->y[bodyIndex + offset],
        //           particles->z[bodyIndex + offset],
        //           particles->mass[bodyIndex + offset],
        //           particles->vx[bodyIndex + offset],
        //           particles->vy[bodyIndex + offset],
        //           particles->vz[bodyIndex + offset],
        //           particles->ax[bodyIndex + offset],
        //           particles->ay[bodyIndex + offset],
        //           particles->az[bodyIndex + offset]);
        //    if (particles->mass[bodyIndex + offset] < 1) {
        //        assert(0);
        //    }
        //}
        // end: debug

        offset += stride;
    }
}

real ExplicitEulerNS::Kernel::Launch::update(Particles *particles, integer n, real dt) {

    ExecutionPolicy executionPolicy;
    return cuda::launch(true, executionPolicy, ::ExplicitEulerNS::Kernel::update, particles, n, dt);

}
