#include <mpi.h>
#include <hip/hip_runtime.h>
#include <fstream>


int main( int argc, char *argv[] )
{
  int rank;
  float *ptr = NULL;
  const size_t elements = 12;
  MPI_Status status;

  float h_array[elements];

  MPI_Init( NULL, NULL );
  MPI_Comm_rank( MPI_COMM_WORLD, &rank );
  hipSetDevice(rank);
  hipMalloc( (void**)&ptr, elements * sizeof(float) );

  for (int i=0; i<elements; i++) {
    if (rank==0) {
      h_array[i] = i;
    }
    else {
      h_array[i] = 0.0;
    }
    printf("[rank %i] h_array[i] = %f\n", rank, i, h_array[i]);    
  }
  
  if (rank==0)
    hipMemcpy(ptr, &h_array, elements*sizeof(float), hipMemcpyHostToDevice);

  if( rank == 0 )
    MPI_Send( ptr, elements, MPI_FLOAT, 1, 0, MPI_COMM_WORLD );
  if( rank == 1 )
    MPI_Recv( ptr, elements, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, &status );

  if (rank == 1) {
    hipMemcpy(&h_array, ptr, elements*sizeof(float), hipMemcpyDeviceToHost);
  }

  if (rank==1)
    printf("Finished communication!\n");

  for (int i=0; i<elements; i++) {
    if (rank == 1) {
      printf("[rank [%i] h_array[i] = %f\n", rank, i, h_array[i]);
    } 
 }

  hipFree( ptr );
  MPI_Finalize();

  return 0;
}
